#include "hip/hip_runtime.h"
#include "CUDAMathematicsFoundationArithmeticCore.cuh"
namespace CUDAMathematicsFoundationArithmeticNamespace
{
	//ComplexAddition��������ӡ�
	__device__ float2 ComplexAddition(float2 Left, float2 Right)
	{
		float2 Result;
		Result.x = Left.x + Right.x;
		Result.y = Left.y + Right.y;
		return Result;
	}

	//ComplexArrayAddition������������ӡ�
	__global__ void ComplexArrayAddition(float2* Left, float2 *Right, int Total_Size)
	{
		int Total_Threads = gridDim.x * blockDim.x;
		int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
		for (int Index = Thread_ID; Index < Total_Size; Index += Total_Threads)
		{
			Left[Index].x = Left[Index].x + Right[Index].x;
			Left[Index].y = Left[Index].y + Right[Index].y;
		}
		__syncthreads();
	}

	//ComplexSubtraction�����������
	__device__ float2 ComplexSubtraction(float2 Left, float2 Right)
	{
		float2 Result;
		Result.x = Left.x - Right.x;
		Result.y = Left.y - Right.y;
		return Result;
	}

	//ComplexArraySubtraction���������������
	__global__ void ComplexArraySubtraction(float2 *Left, float2 *Right, int Total_Size)
	{
		int Total_Threads = gridDim.x * blockDim.x;
		int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
		for (int Index = Thread_ID; Index < Total_Size; Index += Total_Threads)
		{
			Left[Index].x = Left[Index].x - Right[Index].x;
			Left[Index].y = Left[Index].y - Right[Index].y;
		}
		__syncthreads();
	}

	//ComplexScale������������
	__device__ float2 ComplexScale(float Scale, float2 Left)
	{
		float2 Result;
		Result.x = Scale * Left.x;
		Result.y = Scale * Left.y;
		return Result;
	}

	//ComplexMultiplication��������ˡ�
	__device__ float2 ComplexMultiplication(float2 Left, float2 Right)
	{
		float2 Result;
		Result.x = Left.x * Right.x - Left.y * Right.y;
		Result.y = Left.x * Right.y + Left.y * Right.x;
		return Result;
	}

	//ComplexAbsolute�����������ֵ��
	__device__ float ComplexAbsolute(float2 Left)
	{
		float Result;
		Result = sqrt(Left.x * Left.x + Left.y * Left.y);
		return Result;
	}

	//ComplexArrayAbsolute���������������ֵ��
	__global__ void ComplexArrayAbsolute(float *Result, float2 *Left, int Total_Size)
	{
		int Total_Threads = gridDim.x * blockDim.x;
		int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
		for (int Index = Thread_ID; Index < Total_Size; Index += Total_Threads)
		{
			Result[Index] = ComplexAbsolute(Left[Index]);
		}
		__syncthreads();
	}

	//Complex Pointwise Multiplication And Scale�����������˲�������
	__global__ void ComplexPointwiseMultiplicationAndScale(float2 *Left, const float2 *Right, int Total_Size, float Scale)
	{

		int Total_Threads = gridDim.x * blockDim.x;
		int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
		for (int Index = Thread_ID; Index < Total_Size; Index += Total_Threads)
		{
			float2 Complex_Multiplication_Result = ComplexMultiplication(Left[Index], Right[Index]);
			Left[Index] = ComplexScale(Scale, Complex_Multiplication_Result);
		}
		__syncthreads();
	}

	//ComplexConjugate������������㡣
	__global__ void ComplexConjugate(float2 *Left, int Total_Size)
	{
		int Total_Threads = gridDim.x * blockDim.x;
		int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
		for (int Index = Thread_ID; Index < Total_Size; Index += Total_Threads)
		{
			Left[Index].x = Left[Index].x;
			Left[Index].y = -Left[Index].y;
		}
		__syncthreads();
	}

	//FloatArrayAddition��˫float��������ӡ�
	__global__ void FloatArrayAddition(float* Left, float *Right, int Total_Size)
	{
		int Total_Threads = gridDim.x * blockDim.x;
		int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
		for (int Index = Thread_ID; Index < Total_Size; Index += Total_Threads)
		{
			Left[Index] = Left[Index] + Right[Index];
		}
		__syncthreads();
	}

	//FloatArrayScale��float�����������
	__global__ void FloatArrayScale(float *Left, float Scale, int Total_Size)
	{
		int Total_Threads = gridDim.x * blockDim.x;
		int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
		for (int Index = Thread_ID; Index < Total_Size; Index += Total_Threads)
		{
			Left[Index] *= Scale;
		}
		__syncthreads();
	}

	//ComplexArrayScale������������
	__global__ void ComplexArrayScale(float2 *Left, float Scale, int Total_Size)
	{
		int Total_Threads = gridDim.x * blockDim.x;
		int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
		for (int Index = Thread_ID; Index < Total_Size; Index += Total_Threads)
		{
			Left[Index].x *= Scale;
			Left[Index].y *= Scale;
		}
		__syncthreads();
	}

	//RealConvolution��ʵ��������㡣
	__global__ void RealConvolution(float *Result, float *Left, float *Right, int Length_Left, int Length_Right, int Length_Result)
	{
		int Total_Threads = gridDim.x * blockDim.x;
		int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
		for (int Index = Thread_ID; Index < Length_Result; Index += Total_Threads)
		{
			float Temporary_Summation = 0.0f;
			for (int I = 0; I < Length_Right; I++)
			{
				int K = Index - I;
				if (0 <= K && K < Length_Left)
				{
					Temporary_Summation += Left[K] * Right[I];
				}
			}
			Result[Index] = Temporary_Summation;
		}
		__syncthreads();
	}

	//ComplexConvolution������������㡣
	__global__ void ComplexConvolution(float2 *Result, float2 *Left, float2 *Right, int Length_Left, int Length_Right, int Length_Result)
	{
		int Total_Threads = gridDim.x * blockDim.x;
		int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
		for (int Index = Thread_ID; Index < Length_Result; Index += Total_Threads)
		{
			float2 Temporary_Summation;
			Temporary_Summation.x = 0.0f;
			Temporary_Summation.y = 0.0f;
			for (int I = 0; I < Length_Right; I++)
			{
				int K = Index - I;
				if (0 <= K && K < Length_Left)
				{
					Temporary_Summation.x += Left[K].x * Right[I].x;
					Temporary_Summation.y += Left[K].y * Right[I].y;
				}
			}
			Result[Index].x = Result[Index].x + Temporary_Summation.x;
			Result[Index].y = Result[Index].y + Temporary_Summation.y;
		}
		__syncthreads();
	}


	//ComplexConvolution��ʵ��������Ͼ�����㡣
	__global__ void RealComplexConvolution(float2 *Result, float2 *Left, float *Right, int Length_Left, int Length_Right, int Length_Result)
	{
		int Total_Threads = gridDim.x * blockDim.x;
		int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
		for (int Index = Thread_ID; Index < Length_Result; Index += Total_Threads)
		{
			float2 Temporary_Summation;
			Temporary_Summation.x = 0.0f;
			Temporary_Summation.y = 0.0f;
			for (int I = 0; I < Length_Right; I++)
			{
				int K = Index - I;
				if (0 <= K && K < Length_Left)
				{
					Temporary_Summation.x += Left[K].x * Right[I];
					Temporary_Summation.y += Left[K].y * Right[I];
				}
			}
			Result[Index].x = Result[Index].x + Temporary_Summation.x;
			Result[Index].y = Result[Index].y + Temporary_Summation.y;
		}
		__syncthreads();
	}


	//CorrelationCalculation��˫float��������Լ��㡣
	__global__ void CorrelationCalculation(float *Result, float *Left, float *Right, int Result_Length, int Input_Length)
	{
		int Total_Threads = gridDim.x * blockDim.x;
		int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
		for (int Index = Thread_ID; Index < Result_Length; Index += Total_Threads)
		{
			int I = Index - Result_Length / 2;
			for (int J = 0; J < Input_Length; J++)
			{
				int K = I + J;
				if (K >= 0 && K < Input_Length)
				{
					Result[I + Result_Length / 2] += Left[K] * Right[J];
				}
			}
		}
		__syncthreads();
	}

	//TwoFloatArrayScaleAndAddition��˫float������ķ�������ӡ�
	__global__ void TwoFloatArrayScaleAndAddition(float *Result, float *Left, float *Right, float Alpha, float Beta, int Length)
	{
		int Total_Threads = gridDim.x * blockDim.x;
		int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
		for (int Index = Thread_ID; Index < Length; Index += Total_Threads)
		{
			Result[Index] = Left[Index] * Alpha + Right[Index] * Beta;
		}
		__syncthreads();
	}

	//ConvertRealToComplex��ʵ��������ת����
	__global__ void ConvertRealToComplex(float2 *Result, float *Left, int Length)
	{
		int Total_Threads = gridDim.x * blockDim.x;
		int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
		for (int Index = Thread_ID; Index < Length; Index += Total_Threads)
		{
			Result[Index].x = Left[Index];
			Result[Index].y = Left[Index];
		}
		__syncthreads();
	}

	//ConvolutionSame��������㣨sameģʽ����
	__global__ void ConvolutionSame(float* Result, float* Signal_Data, float* Filter_Data, int Data_Length, int Filter_Order)
	{
		int Total_Threads = gridDim.x * blockDim.x;
		int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
		for (int Index = Thread_ID; Index < Data_Length + Filter_Order - 1; Index += Total_Threads)
		{
			Result[Index] = 0;
			for (int I = int(fmaxf(0, Index + 1 - Filter_Order)); I <= int(fminf(Index, Data_Length - 1)); I++)
			{
				Result[Index] += Signal_Data[I] * Filter_Data[Index - I];
			}
		}
		__syncthreads();
	}

	//CUDAExponetialDistributionAssistance��ָ���ֲ�Э����������CUDA��û�з���ָ���ֲ�����������ɺ���������ֻ����Ƹú����������float��ƽ���ֲ������������мӹ����Ӷ����ط���ָ���ֲ���float����������顣
	__global__ void ExponetialDistributionAssistance(float* OutPut_Random_Exponetial_Distribution_Data, float* InPut_Random_Exponetial_Distribution_Data, double lambda, int Total_Size)
	{
		int Total_Threads = gridDim.x * blockDim.x;
		int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
		for (int Index = Thread_ID; Index < Total_Size; Index += Total_Threads)
		{
			if (InPut_Random_Exponetial_Distribution_Data[Index] > 0)
			{
				OutPut_Random_Exponetial_Distribution_Data[Index] = 1.0 - expf(-lambda * InPut_Random_Exponetial_Distribution_Data[Index]);
			}
		}
		__syncthreads();
	}

	//RayleighDistributionAssistance�������ֲ�Э����������CUDA��û�з��������ֲ�����������ɺ���������ֻ����Ƹú����������float��ƽ���ֲ������������мӹ����Ӷ����ط��������ֲ���float����������顣
	__global__ void RayleighDistributionAssistance(float* OutPut_Random_Rayleigh_Distribution_Data, float* InPut_Random_Rayleigh_Distribution_Data, float Rho, int Total_Size)
	{
		int Total_Threads = gridDim.x * blockDim.x;
		int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
		for (int Index = Thread_ID; Index < Total_Size; Index += Total_Threads)
		{
			OutPut_Random_Rayleigh_Distribution_Data[Index] = 1.0 - expf(-(InPut_Random_Rayleigh_Distribution_Data[Index] * InPut_Random_Rayleigh_Distribution_Data[Index]) / (2.0 * Rho * Rho));
		}
		__syncthreads();
	}

	//WeibullDistributionAssistance���������ֲ�Э����������CUDA��û�з����������ֲ�����������ɺ���������ֻ����Ƹú����������float��ƽ���ֲ������������мӹ����Ӷ����ط����������ֲ���float����������顣
	__global__ void WeibullDistributionAssistance(float* OutPut_Random_Weibull_Distribution_Data, float* InPut_Random_Weibull_Distribution_Data, int K, double B, int Total_Size)
	{
		int Total_Threads = gridDim.x * blockDim.x;
		int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
		for (int Index = Thread_ID; Index < Total_Size; Index += Total_Threads)
		{
			OutPut_Random_Weibull_Distribution_Data[Index] = 1.0 - expf(-B * (powf(InPut_Random_Weibull_Distribution_Data[Index], K)));
		}
		__syncthreads();
	}

	__device__ float AntiGamma(float X, float S)
	{
		float  A = 0;
		float Result = 0;
		for (int I = 0; I < 10; I++){
			A += pow(X, I) / tgamma(S + I + 1);
		}
		Result = (pow(X, S)*tgamma(S)*A*exp(-X)) / tgamma(S);
		return Result;
	}

	__global__ void Chi_squareDistributionAssistance(float* OutPut_Random_Chi_Square_Distribution_Data, float* InPut_Random_Chi_Square_Distribution_Data, float K, float X, int Total_Size)
	{
		int Total_Threads = gridDim.x * blockDim.x;
		int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
		float K_2 = K / 2;
		for (int Index = Thread_ID; Index < Total_Size; Index += Total_Threads)
		{
			OutPut_Random_Chi_Square_Distribution_Data[Index] = AntiGamma(InPut_Random_Chi_Square_Distribution_Data[Index] / 2, K_2) / tgamma(K_2);
		}
		__syncthreads();
	}

}