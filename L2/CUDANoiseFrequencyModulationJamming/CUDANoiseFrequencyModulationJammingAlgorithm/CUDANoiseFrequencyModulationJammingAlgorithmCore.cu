#include "hip/hip_runtime.h"
#include "CUDANoiseFrequencyModulationJammingAlgorithmCore.cuh"
#include "../CUDAHeaderFileOfHierarchy1DLL/CUDAMathematicsFoundationArithmeticCore.cuh"
#define  PI_CUDA 3.141592653589793f
__device__ __host__ int CalculationOfOffset(int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int Axis_Y, int Axis_Z)
{
	return Axis_Y * Count_Of_Sampling_Point_Of_Monopulse + Axis_Z * Count_Of_Sampling_Point_Of_Monopulse * Count_Of_Pulse_Group;
}

__device__ __host__ float * GetArrayOneDimensionFloatData(float *Initial_Position, int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int Axis_Y, int Axis_Z)
{
	float * One_Dimension_Float_Data_Position = NULL;
	One_Dimension_Float_Data_Position = Initial_Position + CalculationOfOffset(Count_Of_Sampling_Point_Of_Monopulse, Count_Of_Pulse_Group, Axis_Y, Axis_Z);
	return One_Dimension_Float_Data_Position;
}

__host__ __device__ float2 * GetArrayOneDimensionFloat2Data(float2*Initial_Position, int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int Axis_Y, int Axis_Z)
{
	float2 *One_Dimension_Float2_Data_Position = NULL;
	One_Dimension_Float2_Data_Position = Initial_Position + CalculationOfOffset(Count_Of_Sampling_Point_Of_Monopulse, Count_Of_Pulse_Group, Axis_Y, Axis_Z);
	return One_Dimension_Float2_Data_Position;
}


__global__ void RealFrequencyModulationJammingGenerator(float Amplitude, float Frequency_Modulation_Coefficient, float Medium_Frequency, float Sampling_Frequency_Of_Medium_Frequency, float* Random_Complex, int Count_Of_Sampling_Point_Of_Monopulse, float * Result)
{
	int Total_Threads = gridDim.x * blockDim.x;
	int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int Index = Thread_ID; Index < Count_Of_Sampling_Point_Of_Monopulse; Index += Total_Threads)
	{
		float Temporary_Frequency = Medium_Frequency + Frequency_Modulation_Coefficient * Random_Complex[Index];
		Result[Index] = Result[Index] + Amplitude * cosf(2 * PI_CUDA*Temporary_Frequency*Index / Sampling_Frequency_Of_Medium_Frequency);
	}
}

__global__ void ComplexFrequencyModulationJammingGenerator(float Amplitude, float Frequency_Modulation_Coefficient, float Medium_Frequency, float Sampling_Frequency_Of_Medium_Frequency, float2 * Random_Complex, int Count_Of_Sampling_Point_Of_Monopulse, float2 * Result)
{
	int Total_Threads = gridDim.x * blockDim.x;
	int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;

	for (unsigned int Index = Thread_ID; Index < Count_Of_Sampling_Point_Of_Monopulse; Index += Total_Threads)
	{
		float Frequency_X = Medium_Frequency;
		float Frequency_Y = Medium_Frequency;
		for (int k = 0; k < Index; k++)
		{
			Frequency_X = Frequency_X + Frequency_Modulation_Coefficient*Random_Complex[Index].x;
			Frequency_Y = Frequency_Y + Frequency_Modulation_Coefficient*Random_Complex[Index].x;
		}
		Result[Index].x = Amplitude * cosf(2 * PI_CUDA*Frequency_X * Index / Sampling_Frequency_Of_Medium_Frequency);
		Result[Index].y = Amplitude * sinf(2 * PI_CUDA*Frequency_Y * Index / Sampling_Frequency_Of_Medium_Frequency);
	}
}

float Ino(float X)
{
	float Y, T, E, Z, De, Sde;
	Y = X / 2.0;
	T = 1.0e-08;
	E = 1.0;
	De = 1.0;
	for (int i = 1; i <= 25; i++)
	{
		De = De * Y / i;
		Sde = De * De;
		E = E + Sde;
		if ((E * T) <= Sde)
		{
			continue;
		}
		else
		{
			Z = E;
		}
	}
	return (Z);
}

void KaiserWindow(int Order, double Band_Width, double Alpha, float* hFIR)
{
	float*Hd = (float *)malloc(sizeof(float) * Order);
	memset(Hd, 0.0, (Order)* sizeof(float));
	float* W = (float *)malloc(sizeof(float) * Order);
	memset(W, 0.0, (Order)* sizeof(float));
	int M = 2;
	float A = (Order - 1) / 2;
	float Pi = 4.0*atan(1.0);
	for (int Index = 0; Index < Order; Index++)
	{
		if (Index == A)
		{
			Hd[Index] = Band_Width;
		}
		else
		{
			float B = Index - A;
			Hd[Index] = sin(Pi*B*Band_Width) / (Pi*B);
		}

	}
	//Switch�����ȱ�ݣ��ڸ�ѡ�������case 1��Զѡ������
	switch (M)
	{
		case 1:
		{
			for (int Index = 0; Index < Order; Index++)
			{
				W[Index] = 1.0;
			}
			break;
		}
		case 2:
		{
			float Bes = Ino(Alpha);
			for (int Index = 0; Index < Order; Index++)
			{
				float G = 1.0 - pow(1.0 - 2.0*(float)Index / (float)(Order - 1), 2);
				float Q = Alpha*sqrt(G);
				W[Index] = Ino(Q);
				W[Index] = W[Index] / Bes;
			}
			break;
		}
		default:
			break;
	}
	for (int Index = 0; Index < Order; Index++)
	{
		hFIR[Index] = Hd[Index] * W[Index];
	}
	free(W);
	free(Hd);
}

void BandPassFilter(unsigned int Order, double Start_Frequency, double Stop_Frequency, double Alpha, float *Filter_Pluse_Response)
{
	//ԭ�ȵ�BPF���
	float * Band_pass_Filter = (float *)malloc(sizeof(float) * Order);
	memset(Band_pass_Filter, 0.0, (Order)* sizeof(float));
	float * Start = (float *)malloc(sizeof(float) * Order);
	memset(Start, 0.0, (Order)* sizeof(float));
	float * Stop = (float *)malloc(sizeof(float) * Order);
	memset(Stop, 0.0, (Order)* sizeof(float));
	KaiserWindow(Order, Start_Frequency, Alpha, Start);
	KaiserWindow(Order, Stop_Frequency, Alpha, Stop);
	for (int Index = 0; Index < Order; Index++)
	{
		Band_pass_Filter[Index] = Stop[Index] - Start[Index];
		Filter_Pluse_Response[Index] = Band_pass_Filter[Index];
	}
	free(Band_pass_Filter);
	free(Start);
	free(Stop);
}


void CUDANoiseFrequencyModulationJammingAlgorithmCore(float * Radar_Float_Data_Cube_Address, float2 * Radar_Float2_Data_Cube_Address, int BlockSize, double Amplitude, double Frequency_Modulation_Coefficient, double Jamming_Center_Frequency, double Band_Width, double Sampling_Frequency_Of_Medium_Frequency, double Sampling_Frequency_Of_Band, double PRT, int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int Antenna_Chanel_Count, bool Use_Only_Baseband)
{
	double Beta = 4.55;
	for (int Index_Z = 0; Index_Z < Antenna_Chanel_Count; Index_Z++)
	{
		for (int Index_Y = 0; Index_Y < Count_Of_Pulse_Group; Index_Y++)
		{
			if (!Use_Only_Baseband)
			{
				int GridSize = (Count_Of_Sampling_Point_Of_Monopulse + BlockSize - 1) / BlockSize;
				int Order = 302;
				float * Filter_Pluse_Response = (float *)malloc(sizeof(float) * Order);
				memset(Filter_Pluse_Response, 0.0, (Order) * sizeof(float));
				float Frequency_Begin = (-Band_Width / 2) / Sampling_Frequency_Of_Medium_Frequency * 2.0;
				float Frequency_End = (+Band_Width / 2) / Sampling_Frequency_Of_Medium_Frequency * 2.0;
				BandPassFilter(Order, Frequency_Begin, Frequency_End, Beta, Filter_Pluse_Response);
				float* Filter_Address = NULL;
				checkCudaErrors(hipMalloc((void**)&Filter_Address, Order * sizeof(float)));
				checkCudaErrors(hipMemcpy(Filter_Address, Filter_Pluse_Response, Order * sizeof(float), hipMemcpyHostToDevice));
				float *Noise_Data = NULL;
				checkCudaErrors(hipMalloc((void **)&Noise_Data, Count_Of_Sampling_Point_Of_Monopulse * sizeof(float)));
				checkCudaErrors(hipMemset(Noise_Data, 0, Count_Of_Sampling_Point_Of_Monopulse * sizeof(float)));
				hiprandGenerator_t Random_Generator;
				hiprandStatus_t Curand_Result = hiprandGenerateNormal(Random_Generator, (float*)Noise_Data, Count_Of_Sampling_Point_Of_Monopulse, 0.0, 1.0);
				float * Convolution_Result;//������
				checkCudaErrors(hipMalloc(&Convolution_Result, (Count_Of_Sampling_Point_Of_Monopulse + Order - 1) * sizeof(float)));
				checkCudaErrors(hipMemset(Convolution_Result, 0, (Count_Of_Sampling_Point_Of_Monopulse + Order - 1) * sizeof(float)));
				CUDAMathematicsFoundationArithmeticNamespace::RealConvolution << <GridSize, BlockSize >> >(Convolution_Result, Noise_Data, Filter_Pluse_Response, Count_Of_Sampling_Point_Of_Monopulse, Order, Count_Of_Sampling_Point_Of_Monopulse + Order - 1);
				float * Radar_Signal_Padding = GetArrayOneDimensionFloatData(Radar_Float_Data_Cube_Address, Count_Of_Sampling_Point_Of_Monopulse, Count_Of_Pulse_Group, Index_Y, 0);
				RealFrequencyModulationJammingGenerator << <GridSize, BlockSize >> >(Amplitude, Frequency_Modulation_Coefficient, Jamming_Center_Frequency, Sampling_Frequency_Of_Medium_Frequency, Convolution_Result + Order / 2, Count_Of_Sampling_Point_Of_Monopulse, Radar_Signal_Padding);
				free(Filter_Pluse_Response);
				checkCudaErrors(hipFree(Filter_Address));
				checkCudaErrors(hipFree(Noise_Data));
				checkCudaErrors(hipFree(Convolution_Result));
			
			}
			else
			{
				int GridSize = (Sampling_Frequency_Of_Band * PRT + BlockSize - 1) / BlockSize;
				int Order = 302;
				float * Filter_Pluse_Response = (float *)malloc(sizeof(float) * Order);
				memset(Filter_Pluse_Response, 0.0, (Order)* sizeof(float));
				float Frequency_Begin = (-Band_Width / 2) / Sampling_Frequency_Of_Band * 2.0;
				float Frequency_End = (+Band_Width / 2) / Sampling_Frequency_Of_Band * 2.0;
				BandPassFilter(Order, Frequency_Begin, Frequency_End, Beta, Filter_Pluse_Response);
				float* Filter_Address = NULL;
				checkCudaErrors(hipMalloc((void**)&Filter_Address, Order * sizeof(float)));
				checkCudaErrors(hipMemcpy(Filter_Address, Filter_Pluse_Response, Order * sizeof(float), hipMemcpyHostToDevice));
				float2 *Noise_Data = NULL;
				checkCudaErrors(hipMalloc((void **)&Noise_Data, Count_Of_Sampling_Point_Of_Monopulse * sizeof(float2)));
				checkCudaErrors(hipMemset(Noise_Data, 0, Count_Of_Sampling_Point_Of_Monopulse * sizeof(float2)));
				hiprandGenerator_t Random_Generator;
				hiprandStatus_t Curand_Result = hiprandGenerateNormal(Random_Generator, (float*)Noise_Data, Count_Of_Sampling_Point_Of_Monopulse * 2, 0.0, 1.0);
				float2 * Convolution_Result;//������
				checkCudaErrors(hipMalloc(&Convolution_Result, Count_Of_Sampling_Point_Of_Monopulse * sizeof(float)));
				checkCudaErrors(hipMemset(Convolution_Result, 0, Count_Of_Sampling_Point_Of_Monopulse * sizeof(float)));
				CUDAMathematicsFoundationArithmeticNamespace::RealComplexConvolution << <GridSize, BlockSize >> >(Convolution_Result, Noise_Data, Filter_Pluse_Response, Count_Of_Sampling_Point_Of_Monopulse, Order, Count_Of_Sampling_Point_Of_Monopulse);
				float2 * Radar_Signal_Padding = GetArrayOneDimensionFloat2Data(Radar_Float2_Data_Cube_Address, Count_Of_Sampling_Point_Of_Monopulse, Count_Of_Pulse_Group, Index_Y, Index_Z);
				ComplexFrequencyModulationJammingGenerator << <GridSize, BlockSize >> >(Amplitude, Frequency_Modulation_Coefficient, Jamming_Center_Frequency, Sampling_Frequency_Of_Medium_Frequency, Convolution_Result, Count_Of_Sampling_Point_Of_Monopulse, Radar_Signal_Padding);
				free(Filter_Pluse_Response);
				checkCudaErrors(hipFree(Filter_Address));
				checkCudaErrors(hipFree(Noise_Data));
				checkCudaErrors(hipFree(Convolution_Result));
			}
		}
	}
}