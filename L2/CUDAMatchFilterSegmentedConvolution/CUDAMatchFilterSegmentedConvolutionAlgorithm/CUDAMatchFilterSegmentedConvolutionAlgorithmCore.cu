#include "hip/hip_runtime.h"
#include "CUDAMatchFilterSegmentedConvolutionAlgorithmCore.cuh"
#include "../CUDAHeaderFileOfHierarchy1DLL/CUDAMathematicsFoundationArithmeticCore.cuh"
int CalculateSectionLength(int N) 
{
	// calculate section length M that M + N - 1 is powers of 2 and M is close to N
	int fFFT_Length = 1;
	while ((2 * N) > fFFT_Length)
	{
		fFFT_Length *= 2;
	}
	return fFFT_Length + 1 - N;
}

int CalculatePaddedLength(int M, int Input_Data_Size)
{
	// calculate the padded length of long signal that the padded length is multiplies of M
	int fFFT_Length = 0;
	while (fFFT_Length < Input_Data_Size)
	{
		fFFT_Length += M;
	}
	return fFFT_Length;
}

/*
CORE FUNCTIONS that perform segmented convolution using overlap-add/overlap-save method
Four versions of segmented convolution include: overlap-add, in-place version of overlap-add
(without creating a buffer to store output), overlap-save, in-place version of overlap-save
*/
/*
The following comments of the use of parameters apply to all four versions
of segmented convolution function
@param [in] Plan
cufft plan used for fft operations of each segmented convolution ÿһ�ֶξ��FFT������
@param [in] Input_Long_Signal
input array of long signal to be conducted convolution ���зֶξ���ĳ��źŵ�ַ
@param [in] Input_Short_Signal
input array of short signal to be conducted convolution ���зֶξ���Ķ��źŵ�ַ
@param [out] y
output array that stores convolution result �洢�������ĵ�ַ
@param [in] Input_Long_Signal_Size
size of data elements of long signal (after zero-padding to be multiplies of M) ���ź�����Ԫ�س���
@param [in] y_len
size of data elements of output array �������������Ԫ�س���
@param [in] M
size of each section for sectioned convolution �ֶξ��ÿһ������Ԫ�س���
@param [in] N
size of data elements for short signal ���ź�����Ԫ�س���
@param [in] scaleFactor
parameter used in ComplexPointwiseMulAndScale kernel function
*/
void ConvolutionOverlapSaveInPlace(hipfftHandle Plan, hipfftComplex *Input_Long_Signal, hipfftComplex *Input_Short_Signal, int Input_Long_Signal_Size, int M, int N, float Scale_Factor)
{
	// allocate GPU memory for buffer
	hipfftComplex *Buffer;
	checkCudaErrors(hipMalloc(&Buffer, (M + N - 1) * sizeof(hipfftComplex)));
	// copy the first section of x to buffer
	// set the first N - 1 elements to 0
	checkCudaErrors(hipMemset(Buffer, 0, (N - 1) * sizeof(hipfftComplex)));
	checkCudaErrors(hipMemcpy(Buffer + N - 1, Input_Long_Signal, M * sizeof(hipfftComplex), hipMemcpyDeviceToDevice));
	// set block parameters
	// ALERT: assume N + M - 1 is multiplies of 32 (launch integral warps)
	int BlockSize = ((N + M - 1) > 512) ? 512 : N + M - 1;
	int GridSize = ((N + M - 1) % BlockSize == 0) ? (N + M - 1) / BlockSize : (N + M - 1) / BlockSize + 1;
	int Start = -N + 1;
	int End = Input_Long_Signal_Size;
	int Stride = M;
	// partition array x into M-length sub-arrays (in overlap-save fashion) and perform convolution separately
	hipfftResult Result;
	for (int Index = Start; Index < End; Index += Stride)
	{

		// perform FFT operation on buffer (which stores each section of x per iteration)
		Result = hipfftExecC2C(Plan, Buffer, Buffer, HIPFFT_FORWARD);
		if (Result != HIPFFT_SUCCESS) {
			printf("Error with in-place FFT operation on section of long signal. cuFFT Error code in %dth iteration: %d\n", Index, Result);
			return;
		}

		// perform convolution on each section
		CUDAMathematicsFoundationArithmeticNamespace::ComplexPointwiseMultiplicationAndScale << <GridSize, BlockSize >> >(Buffer, Input_Short_Signal, N + M - 1, Scale_Factor);

		// perform IFFT operation on each section
		Result = hipfftExecC2C(Plan, Buffer, Buffer, HIPFFT_BACKWARD);
		if (Result != HIPFFT_SUCCESS)
		{
			printf("Error with in-place IFFT operation on section of long signal. cuFFT Error code in %dth iteration: %d\n", Index, Result);
			return;
		}

		// copy partial array of x (next section) to buffer
		checkCudaErrors(hipMemcpy(Buffer, Input_Long_Signal + Index + M, (M + N - 1) * sizeof(hipfftComplex), hipMemcpyDeviceToDevice));

		// copy partial Result to output array
		if ((Index + M + N - 1) >= (Input_Long_Signal_Size + N - 1))
		{
			checkCudaErrors(hipMemcpy(Input_Long_Signal + Index + N - 1, Buffer + N - 1, (Input_Long_Signal_Size - Index) * sizeof(hipfftComplex), hipMemcpyDeviceToDevice));
		}
		else
		{
			checkCudaErrors(hipMemcpy(Input_Long_Signal + Index + N - 1, Buffer + N - 1, M * sizeof(hipfftComplex), hipMemcpyDeviceToDevice));
		}
	}
	// clean buffer data
	checkCudaErrors(hipFree(Buffer));
}


void ConvolutionOverlapSave(hipfftHandle Plan, hipfftComplex *Input_Long_Signal, hipfftComplex *Input_Short_Signal, hipfftComplex *Output_Signal, int Input_Long_Signal_Size, int Output_Signal_Size, int M, int N, float Scale_Factor)
{
	// allocate GPU memory for buffers
	hipfftComplex *Buffer;
	checkCudaErrors(hipMalloc(&Buffer, (M + N - 1) * sizeof(hipfftComplex)));
	// set block parameters
	// ALERT: assume N + M - 1 is multiplies of 32 (launch integral warps)
	unsigned BlockSize = ((N + M - 1) > 512) ? 512 : N + M - 1;
	unsigned GridSize = ((N + M - 1) % BlockSize == 0) ? (N + M - 1) / BlockSize : ((N + M - 1) / BlockSize + 1);
	int Start = -N + 1;
	int End = Input_Long_Signal_Size;
	int Stride = M;
	// partition array x into M-length sub-arrays (in overlap-save fashion) and perform convolution separately
	hipfftResult Result;
	for (int Index = Start; Index < End; Index += Stride)
	{
		// copy partial array of x to buffer
		if (Index < 0)
		{
			// set first N - 1 elements to zero for the first section
			checkCudaErrors(hipMemset(Buffer, 0, (N - 1) * sizeof(hipfftComplex)));
			// copy the last M elements to the buffer
			checkCudaErrors(hipMemcpy(Buffer + N - 1, Input_Long_Signal, M * sizeof(hipfftComplex), hipMemcpyDeviceToDevice));
		}
		else if ((Index + M + N - 1) >= Input_Long_Signal_Size)
		{
			// set the buffer to zero for the last section
			checkCudaErrors(hipMemset(Buffer, 0, (M + N - 1) * sizeof(hipfftComplex)));
			// copy in-array elements to the buffer
			checkCudaErrors(hipMemcpy(Buffer, Input_Long_Signal + Index, (Input_Long_Signal_Size - Index) * sizeof(hipfftComplex), hipMemcpyDeviceToDevice));
		}
		else
		{
			checkCudaErrors(hipMemcpy(Buffer, Input_Long_Signal + Index, (M + N - 1) * sizeof(hipfftComplex), hipMemcpyDeviceToDevice));
		}
		// perform FFT operation on each section
		Result = hipfftExecC2C(Plan, Buffer, Buffer, HIPFFT_FORWARD);
		if (Result != HIPFFT_SUCCESS) 
		{
			printf("Error with in-place FFT operation on section of long signal. cuFFT Error code in %dth iteration: %d\n", Index, Result);
			return;
		}
		// perform convolution on each section
		CUDAMathematicsFoundationArithmeticNamespace::ComplexPointwiseMultiplicationAndScale << <GridSize, BlockSize >> >(Buffer, Input_Short_Signal, N + M - 1, Scale_Factor);
		// perform IFFT operation on each section
		Result = hipfftExecC2C(Plan, Buffer, Buffer, HIPFFT_BACKWARD);
		if (Result != HIPFFT_SUCCESS)
		{
			printf("Error with in-place IFFT operation on section of long signal. cuFFT Error code in %dth iteration: %d\n", Index, Result);
			return;
		}

		// copy partial Result to output array
		if ((Index + M + N - 1) >= Output_Signal_Size)
		{
			checkCudaErrors(hipMemcpy(Output_Signal + Index + N - 1, Buffer + N - 1, (Output_Signal_Size - Index - N + 1) * sizeof(hipfftComplex), hipMemcpyDeviceToDevice));
		}
		else
		{
			checkCudaErrors(hipMemcpy(Output_Signal + Index + N - 1, Buffer + N - 1, M * sizeof(hipfftComplex), hipMemcpyDeviceToDevice));
		}
	}
	// clean buffer data
	checkCudaErrors(hipFree(Buffer));
}


void ConvolutionOverlapAdd(hipfftHandle Plan, hipfftComplex *Input_Long_Signal, hipfftComplex *Input_Short_Signal, hipfftComplex *Output_Signal, int Input_Long_Signal_Size, int Output_Signal_Size, int M, int N, float Scale_Factor)
{
	// allocate GPU memory for buffer
	hipfftComplex *Buffer;
	checkCudaErrors(hipMalloc(&Buffer, (N + M - 1) * sizeof(hipfftComplex)));
	// set output array values to zero
	checkCudaErrors(hipMemset(Output_Signal, 0, Output_Signal_Size * sizeof(hipfftComplex)));
	// set block parameters
	// ALERT: assume N + M - 1 is multiplies of 32 (launch integral warps)
	int BlockSize = ((N + M - 1) > 512) ? 512 : N + M - 1;
	int GridSize = ((N + M - 1) % BlockSize == 0) ? (N + M - 1) / BlockSize : (N + M - 1) / BlockSize + 1;
	// partition array x into M-length sub-arrays and perform convolution separately
	hipfftResult Result;
	for (int Index = 0; Index < Input_Long_Signal_Size; Index += M)
	{
		// clear the buffer before fft operation per iteration
		// copy each section from input signal to buffer
		checkCudaErrors(hipMemset(Buffer, 0, (M + N - 1) * sizeof(hipfftComplex)));
		checkCudaErrors(hipMemcpy(Buffer, Input_Long_Signal + Index, M * sizeof(hipfftComplex), hipMemcpyDeviceToDevice));
		// perform FFT operation on each section
		Result = hipfftExecC2C(Plan, Buffer, Buffer, HIPFFT_FORWARD);
		if (Result != HIPFFT_SUCCESS)
		{
			printf("Error with in-place FFT operation on section of long signal. cuFFT Error code in %dth iteration: %d\n", Index, Result);
			return;
		}
		// perform convolution on each section
		CUDAMathematicsFoundationArithmeticNamespace::ComplexPointwiseMultiplicationAndScale << <GridSize, BlockSize >> >(Buffer, Input_Short_Signal, N + M - 1, Scale_Factor);
		// perform IFFT operation on each section
		Result = hipfftExecC2C(Plan, Buffer, Buffer, HIPFFT_BACKWARD);
		if (Result != HIPFFT_SUCCESS)
		{
			printf("Error with in-place IFFT operation on section of long signal. cuFFT Error code in %dth iteration: %d\n", Index, Result);
			return;
		}
		// add partial Result (N + M - 1 elements) to the output array
		// perform overlap-add
		CUDAMathematicsFoundationArithmeticNamespace::ComplexArrayAddition << <GridSize, BlockSize >> >(Output_Signal + Index, Buffer, N + M - 1);

	}
	// clean buffer data
	checkCudaErrors(hipFree(Buffer));
}


void ConvolutionOverlapAddInPlace(hipfftHandle Plan, hipfftComplex *Input_Long_Signal, hipfftComplex *Input_Short_Signal, int Input_Long_Signal_Size, int M, int N, float Scale_Factor)
{
	// allocate GPU memory for buffer
	hipfftComplex *Buffer;
	checkCudaErrors(hipMalloc(&Buffer, (N + M - 1) * sizeof(hipfftComplex)));
	// clear the buffer and copy the first M-length section of input array to the buffer
	checkCudaErrors(hipMemset(Buffer, 0, (M + N - 1) * sizeof(hipfftComplex)));
	checkCudaErrors(hipMemcpy(Buffer, Input_Long_Signal, M * sizeof(hipfftComplex), hipMemcpyDeviceToDevice));
	// set the first M-length section of input array to 0
	checkCudaErrors(hipMemset(Input_Long_Signal, 0, M*sizeof(hipfftComplex)));
	// set block parameters
	// ALERT: assume N + M - 1 is multiplies of 32 (launch integral warps)
	int BlockSize = ((N + M - 1) > 512) ? 512 : N + M - 1;
	int GridSize = ((N + M - 1) % BlockSize == 0) ? (N + M - 1) / BlockSize : ((N + M - 1) / BlockSize + 1);
	// partition array x into M-length sub-arrays and perform convolution separately
	hipfftResult Result;
	for (int Index = 0; Index < Input_Long_Signal_Size; Index += M)
	{

		// perform FFT operation on each section
		Result = hipfftExecC2C(Plan, Buffer, Buffer, HIPFFT_FORWARD);
		if (Result != HIPFFT_SUCCESS)
		{
			printf("Error with in-place FFT operation on section of long signal. cuFFT Error code in %dth iteration: %d\n", Index, Result);
			return;
		}
		// perform convolution on each section
		CUDAMathematicsFoundationArithmeticNamespace::ComplexPointwiseMultiplicationAndScale << <GridSize, BlockSize >> >(Buffer, Input_Short_Signal, N + M - 1, Scale_Factor);
		// perform IFFT operation on each section
		Result = hipfftExecC2C(Plan, Buffer, Buffer, HIPFFT_BACKWARD);
		if (Result != HIPFFT_SUCCESS)
		{
			printf("Error with in-place IFFT operation on section of long signal. cuFFT Error code in %dth iteration: %d\n", Index, Result);
			return;
		}
		// perform boundary check
		if ((Index + M) < Input_Long_Signal_Size)
		{
			// clear the buffer before fft operation per iteration
			// copy next section from input array to buffer
			checkCudaErrors(hipMemset(Buffer, 0, (M + N - 1) * sizeof(hipfftComplex)));
			checkCudaErrors(hipMemcpy(Buffer, Input_Long_Signal + Index + M, M * sizeof(hipfftComplex), hipMemcpyDeviceToDevice));
			// set next section of input array to 0
			checkCudaErrors(hipMemset(Input_Long_Signal + Index + M, 0, M * sizeof(hipfftComplex)));
		}
		// add partial Result (N + M - 1 elements) to the output array
		// perform overlap-add
		CUDAMathematicsFoundationArithmeticNamespace::ComplexArrayAddition << <GridSize, BlockSize >> >(Input_Long_Signal + Index, Buffer, N + M - 1);

	}
	checkCudaErrors(hipFree(Buffer));
}



void CUDAMatchFilterSegmentedConvolutionAlgorithmCore(hipfftHandle Plan, float2* Signal_Padded_G, float2 *Kernel_Padded_Result_G,  int Kenel_Padded_Size,  int Pulse_Size, float Scale_Factor)
{
	//paddedsize����ʱ��ά���źŵĳ��ȣ���0���ֵ
	//kenel_padded_size����Ч�ķ����ź������ȵĳ��ȣ����źţ�ʵ����Ӧ���������ȶ�Ӧ�Ĳ�����
	//pulsesize����ʱ��ά���źŵĳ��ȣ�û�в�0��ֵ

	// calculate the section length for input longer signal and output array
	int IsectionLength = CalculateSectionLength(Kenel_Padded_Size);
	// calculate the approximate length of input signal that equals to multiplies of IsectionLength 
	int Signal_padded_G_padded_len = CalculatePaddedLength(IsectionLength, Pulse_Size);
	// perform FFT operation on kernel_padded_result_G_fft
	hipfftResult Result = hipfftExecC2C(Plan, Kernel_Padded_Result_G, Kernel_Padded_Result_G, HIPFFT_FORWARD);
	if (Result != HIPFFT_SUCCESS)
	{  // check if cufft plan is successfully destroyed
		printf("Error with in-place FFT operation on short signal. cuFFT Error code: %d\n", Result);
		return;
	}

	// uncomment the code if using out-of-place version of segmented convolution functions
	// create a buffer that stores Signal_Padded_G data and perform zero-padding
	/*
	hipfftComplex* signal_padded_G_copy;
	unsigned signal_padded_G_copy_len = calcPaddedLength(IsectionLength, Pulse_Size);
	checkCudaErrors(hipMalloc(&signal_padded_G_copy, signal_padded_G_copy_len*sizeof(hipfftComplex)));
	checkCudaErrors(hipMemset(signal_padded_G_copy, 0, signal_padded_G_copy_len *sizeof(hipfftComplex)));
	checkCudaErrors(hipMemcpy(signal_padded_G_copy, signal_padded_G, Pulse_Size*sizeof(hipfftComplex), hipMemcpyDeviceToDevice));
	*/

	// perform sectioned convolution using overlap-add/overlap-save
	/*
	ConvolutionOverlapAdd(plan, signal_padded_G_copy, Kernel_Padded_Result_G, (hipfftComplex *)signal_padded_G, signal_padded_G_copy_len, signal_padded_G_copy_len + Kenel_Padded_Size - 1, isectionLength, Kenel_Padded_Size, Scale_Factor);
	ConvolutionOverlapSave(plan, signal_padded_G_copy, Kernel_Padded_Result_G, (hipfftComplex *)signal_padded_G, signal_padded_G_copy_len, signal_padded_G_copy_len + Kenel_Padded_Size - 1, isectionLength, Kenel_Padded_Size, Scale_Factor);
	*/
	ConvolutionOverlapAddInPlace(Plan, Signal_Padded_G, Kernel_Padded_Result_G, Signal_padded_G_padded_len, IsectionLength, Kenel_Padded_Size, Scale_Factor);
	/*
	ConvolutionOverlapSaveInPlace(Plan, Signal_Padded_G, Kernel_Padded_Result_G, Signal_padded_G_padded_len, IsectionLength, Kenel_Padded_Size, Scale_Factor);
	*/
}
