#include "hip/hip_runtime.h"
#include "CUDAMatchFilterAlgorithmCore.cuh"
#include "../CUDAHeaderFileOfHierarchy1DLL/CUDAMathematicsFoundationArithmeticCore.cuh"
void CUDAMatchFilterAlgorithmCore(hipfftHandle Plan_Forward, hipfftHandle Plan_Backword, int BlockSize, int Baseband_Total_Size, int ScaleFactor, float2 * Baseband_Signal, float2 * Baseband_Frequency_Domain_Signal_Kernel)
{
	checkCudaErrors(hipfftExecC2C(Plan_Forward, (hipfftComplex *)Baseband_Signal, (hipfftComplex *)Baseband_Signal, HIPFFT_FORWARD));
	unsigned int GridSize = (Baseband_Total_Size + BlockSize - 1) / BlockSize;
	//ʹ�û������������ݡ�
	CUDAMathematicsFoundationArithmeticNamespace::ComplexPointwiseMultiplicationAndScale << <GridSize, BlockSize >> >(Baseband_Signal, Baseband_Frequency_Domain_Signal_Kernel, BlockSize, ScaleFactor);
	checkCudaErrors(hipfftExecC2C(Plan_Backword, (hipfftComplex *)Baseband_Signal, (hipfftComplex *)Baseband_Signal, HIPFFT_BACKWARD));

}