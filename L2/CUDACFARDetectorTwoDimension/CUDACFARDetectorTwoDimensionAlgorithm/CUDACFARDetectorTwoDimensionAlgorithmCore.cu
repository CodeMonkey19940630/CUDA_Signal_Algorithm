#include "hip/hip_runtime.h"
#include "CUDACFARDetectorTwoDimensionAlgorithmCore.cuh"
#include "../CUDAHeaderFileOfHierarchy1DLL/CUDAMathematicsFoundationArithmeticCore.cuh"

__host__ __device__ int CalculationOfOffset(int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int Axis_Z)
{
	return Axis_Z * Count_Of_Sampling_Point_Of_Monopulse * Count_Of_Pulse_Group;
}

__host__ __device__ float2 * GetArrayTwoDimensionFloat2Data(float2*Initial_Position, int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int Axis_Z)
{
	float2 *Two_Dimension_Float2_Data_Position = NULL;
	Two_Dimension_Float2_Data_Position = Initial_Position + CalculationOfOffset(Count_Of_Sampling_Point_Of_Monopulse, Count_Of_Pulse_Group, Axis_Z);
	return Two_Dimension_Float2_Data_Position;
}

__global__ void CACFAR2D(float * CFAR_Reference, float * Signal_Absolute, int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int GuardCell_F, int WindowCell_F, int GuardCell_S, int WindowCell_S)
{
	int Total_Threads = gridDim.x * blockDim.x;
	int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
	const  int Total_Size = Count_Of_Sampling_Point_Of_Monopulse * Count_Of_Pulse_Group;

	int Total_Window_Size = (2 * (WindowCell_F + GuardCell_F) + 1) * (2 * (WindowCell_S + GuardCell_S) + 1);
	int Guard_Window_Size = (2 * GuardCell_F + 1)*(2 * GuardCell_S + 1);
	int Reference_Window_Size = Total_Window_Size - Guard_Window_Size;

	for (int Index = Thread_ID; Index < Total_Size; Index += Total_Threads)
	{
		CFAR_Reference[Index] = 0.0f;
		int Axis_Y = Index / Count_Of_Sampling_Point_Of_Monopulse;
		int Axis_X = Index - Axis_Y * Count_Of_Sampling_Point_Of_Monopulse;

		//�°汾������ʽ
		if ((Axis_X < WindowCell_F + GuardCell_F || Axis_X >= Count_Of_Sampling_Point_Of_Monopulse - 1 - (WindowCell_F + GuardCell_F - 1)) || (Axis_Y < WindowCell_S + GuardCell_S || Axis_Y >= Count_Of_Pulse_Group - 1 - (WindowCell_S + GuardCell_S - 1)))
		{
			continue;
		}
		else
		{
			//���ž��뷽�����
			float Sum = 0.0;
			for (int X = Axis_X - GuardCell_F - WindowCell_F; X <= Axis_X + GuardCell_F + WindowCell_F; X++)
			{
				for (int Y = Axis_Y - GuardCell_S - WindowCell_S; Y <= Axis_Y + GuardCell_S + WindowCell_S; Y++)
				{
					int Index = X + Y * Count_Of_Sampling_Point_Of_Monopulse;
					if ((X >= Axis_X - GuardCell_F && X <= Axis_X + GuardCell_F) && (Y >= Axis_Y - GuardCell_S && Y <= Axis_Y + GuardCell_S))
					{
						continue;
					}

					Sum += Signal_Absolute[Index];

				}
			}

			CFAR_Reference[Index] = Sum / Reference_Window_Size;

		}
	}
	__syncthreads();
}

__global__ void SOCACFAR2D(float *CFAR_Reference, float *Signal_Absolute, int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int GuardCell_F, int WindowCell_F, int GuardCell_S, int WindowCell_S)
{
	int Total_Threads = gridDim.x * blockDim.x;
	int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
	const  int Total_Size = Count_Of_Sampling_Point_Of_Monopulse*Count_Of_Pulse_Group;

	int Total_Window_Size = (2 * (WindowCell_F + GuardCell_F) + 1) * (2 * (WindowCell_S + GuardCell_S) + 1);
	int Guard_Window_Size = (2 * GuardCell_F + 1)*(2 * GuardCell_S + 1);
	int Reference_Window_Size = Total_Window_Size - Guard_Window_Size;

	for (int Index = Thread_ID; Index < Total_Size; Index += Total_Threads)
	{
		CFAR_Reference[Index] = 0.0f;
		int Axis_Y = Index / Count_Of_Sampling_Point_Of_Monopulse;
		int Axis_X = Index - Axis_Y*Count_Of_Sampling_Point_Of_Monopulse;

		//�°汾������ʽ
		if ((Axis_X < WindowCell_F + GuardCell_F || Axis_X >= Count_Of_Sampling_Point_Of_Monopulse - 1 - (WindowCell_F + GuardCell_F - 1)) ||
			(Axis_Y < WindowCell_S + GuardCell_S || Axis_Y >= Count_Of_Pulse_Group - 1 - (WindowCell_S + GuardCell_S - 1)))
		{
			continue;
		}
		else
		{
			//���ž��뷽�����
			float Left_Sum = 0.0;
			float Right_Sum = 0.0;
			for (int X = Axis_X - GuardCell_F - WindowCell_F; X <= Axis_X + GuardCell_F + WindowCell_F; X++)
			{
				for (int Y = Axis_Y - GuardCell_S - WindowCell_S; Y <= Axis_Y + GuardCell_S + WindowCell_S; Y++)
				{
					int Index = X + Y * Count_Of_Sampling_Point_Of_Monopulse;
					if ((X >= Axis_X - GuardCell_F && X <= Axis_X + GuardCell_F) && (Y >= Axis_Y - GuardCell_S && Y <= Axis_Y + GuardCell_S))
					{
						continue;
					}

					if (X < Axis_X - GuardCell_F)
					{
						Left_Sum += Signal_Absolute[Index];
					}

					if (X >= Axis_X + GuardCell_F + 1)
					{
						Right_Sum += Signal_Absolute[Index];
					}

				}
			}

			if (Left_Sum < Right_Sum)
			{
				CFAR_Reference[Index] = Left_Sum * 1.0 / (Reference_Window_Size / 2.0);
			}
			else
			{
				CFAR_Reference[Index] = Right_Sum * 1.0 / (Reference_Window_Size / 2.0);
			}

		}
	}
	__syncthreads();
}

__global__ void GOCACFAR2D(float *CFAR_Reference, float *Signal_Absolute, int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int GuardCell_F, int WindowCell_F, int GuardCell_S, int WindowCell_S)
{
	int Total_Threads = gridDim.x * blockDim.x;
	int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
	const  int Total_Size = Count_Of_Sampling_Point_Of_Monopulse*Count_Of_Pulse_Group;

	int Total_Window_Size = (2 * (WindowCell_F + GuardCell_F) + 1) * (2 * (WindowCell_S + GuardCell_S) + 1);
	int Guard_Window_Size = (2 * GuardCell_F + 1)*(2 * GuardCell_S + 1);
	int Reference_Window_Size = Total_Window_Size - Guard_Window_Size;

	for (int Index = Thread_ID; Index < Total_Size; Index += Total_Threads)
	{
		CFAR_Reference[Index] = 0.0f;
		int Axis_Y = Index / Count_Of_Sampling_Point_Of_Monopulse;
		int Axis_X = Index - Axis_Y*Count_Of_Sampling_Point_Of_Monopulse;

		//�°汾������ʽ
		if ((Axis_X < WindowCell_F + GuardCell_F || Axis_X >= Count_Of_Sampling_Point_Of_Monopulse - 1 - (WindowCell_F + GuardCell_F - 1)) ||
			(Axis_Y < WindowCell_S + GuardCell_S || Axis_Y >= Count_Of_Pulse_Group - 1 - (WindowCell_S + GuardCell_S - 1)))
		{
			continue;
		}
		else
		{
			//���ž��뷽�����
			float Left_Sum = 0.0;
			float Right_Sum = 0.0;
			for (int X = Axis_X - GuardCell_F - WindowCell_F; X <= Axis_X + GuardCell_F + WindowCell_F; X++)
			{
				for (int Y = Axis_Y - GuardCell_S - WindowCell_S; Y <= Axis_Y + GuardCell_S + WindowCell_S; Y++)
				{
					int Index = X + Y*Count_Of_Sampling_Point_Of_Monopulse;
					if ((X >= Axis_X - GuardCell_F && X <= Axis_X + GuardCell_F) && (Y >= Axis_Y - GuardCell_S && Y <= Axis_Y + GuardCell_S))
					{
						continue;
					}

					if (X < Axis_X - GuardCell_F)
					{
						Left_Sum += Signal_Absolute[Index];
					}

					if (X >= Axis_X + GuardCell_F + 1)
					{
						Right_Sum += Signal_Absolute[Index];
					}

				}
			}

			if (Left_Sum < Right_Sum)
			{
				CFAR_Reference[Index] = Right_Sum * 1.0 / (Reference_Window_Size / 2.0);
			}
			else
			{
				CFAR_Reference[Index] = Left_Sum * 1.0 / (Reference_Window_Size / 2.0);
			}

		}
	}
	__syncthreads();
}

__global__ void DetectByCFAR2D(bool *Dectect_Result, float *Signal_Absolute, float *CFAR_Reference, int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int GuardCell_F, int WindowCell_F, int GuardCell_S, int WindowCell_S, float Min_Detect_Factor, float Min_Detect_Threshold)
{
	int Total_Threads = gridDim.x * blockDim.x;
	int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
	const  int Total_Size = Count_Of_Sampling_Point_Of_Monopulse * Count_Of_Pulse_Group;
	for (int Index = Thread_ID; Index < Total_Size; Index += Total_Threads)
	{
		Dectect_Result[Index] = false;

		int Axis_Y = Index / Count_Of_Sampling_Point_Of_Monopulse;
		int Axis_X = Index - Axis_Y*Count_Of_Sampling_Point_Of_Monopulse;

		if ((Axis_X < WindowCell_F + GuardCell_F || Axis_X >= Count_Of_Sampling_Point_Of_Monopulse - 1 - (WindowCell_F + GuardCell_F - 1)) || (Axis_Y < WindowCell_S + GuardCell_S || Axis_Y >= Count_Of_Pulse_Group - 1 - (WindowCell_S + GuardCell_S - 1)))
		{
			continue;
		}
		Dectect_Result[Index] = (Signal_Absolute[Index] >(CFAR_Reference[Index] * Min_Detect_Factor)) && (Signal_Absolute[Index] > Min_Detect_Threshold);
	}
	__syncthreads();
}

void CUDACFARDetectorTwoDimensionAlgorithmCore(float2 * Radar_Data_Cube_Address, float * CFAR_Absolute_Detected_Data, float * CFAR_Reference, bool * CFAR_Result, int BlockSize, int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int Antenna_Chanel_Count, int GuardCell_F, int WindowCell_F, int GuardCell_S, int WindowCell_S, double Min_Detect_Factor, double Min_Detect_Threshold, int CFAR_Method_Select)
{
	unsigned int GridSize = (Count_Of_Sampling_Point_Of_Monopulse + BlockSize - 1) / BlockSize;
	for (int Index_Z = 0; Index_Z < Antenna_Chanel_Count; Index_Z++)
	{
		float2 * Radar_Signal_Padding = GetArrayTwoDimensionFloat2Data(Radar_Data_Cube_Address, Count_Of_Sampling_Point_Of_Monopulse, Count_Of_Pulse_Group, Index_Z);
		int Offset = Index_Z * Count_Of_Sampling_Point_Of_Monopulse * Count_Of_Pulse_Group;
		int Count_Of_Sampling_Point_Of_CPI = Count_Of_Sampling_Point_Of_Monopulse * Count_Of_Pulse_Group;
		float * CFAR_Signal_Padding = CFAR_Absolute_Detected_Data + Offset;
		float *CFAR_Reference_Padding = CFAR_Reference + Offset;
		bool *Detect_Result = CFAR_Result + Offset;
		CUDAMathematicsFoundationArithmeticNamespace::ComplexArrayAbsolute << <GridSize, BlockSize >> >(CFAR_Signal_Padding, Radar_Signal_Padding, Count_Of_Sampling_Point_Of_CPI);
		switch (CFAR_Method_Select)
		{
		case 0:
		{
			CACFAR2D << <GridSize, BlockSize >> >(CFAR_Reference_Padding, CFAR_Signal_Padding, Count_Of_Sampling_Point_Of_Monopulse, Count_Of_Pulse_Group, GuardCell_F, WindowCell_F, GuardCell_S, WindowCell_S);
			break;
		}
		case 1:
		{
			SOCACFAR2D << <GridSize, BlockSize >> >(CFAR_Reference_Padding, CFAR_Signal_Padding, Count_Of_Sampling_Point_Of_Monopulse, Count_Of_Pulse_Group, GuardCell_F, WindowCell_F, GuardCell_S, WindowCell_S);
			break;
		}
		case 2:
		{
			GOCACFAR2D << <GridSize, BlockSize >> >(CFAR_Reference_Padding, CFAR_Signal_Padding, Count_Of_Sampling_Point_Of_Monopulse, Count_Of_Pulse_Group, GuardCell_F, WindowCell_F, GuardCell_S, WindowCell_S);
			break;
		}
		default:
		{
			CACFAR2D << <GridSize, BlockSize >> >(CFAR_Reference_Padding, CFAR_Signal_Padding, Count_Of_Sampling_Point_Of_Monopulse, Count_Of_Pulse_Group, GuardCell_F, WindowCell_F, GuardCell_S, WindowCell_S);
			break;
		}
		}
		DetectByCFAR2D << <GridSize, BlockSize >> >(Detect_Result, CFAR_Signal_Padding, CFAR_Reference_Padding, Count_Of_Sampling_Point_Of_Monopulse, Count_Of_Pulse_Group, GuardCell_F, WindowCell_F, GuardCell_S, WindowCell_S, Min_Detect_Factor, Min_Detect_Threshold);
	}
}