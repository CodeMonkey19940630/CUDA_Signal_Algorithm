#include "hip/hip_runtime.h"
#include "CUDAPulseDopplerRadarDisplayAlgorithmCore.cuh"
#include "../CUDAHeaderFileOfHierarchy1DLL/CUDAMathematicsFoundationArithmeticCore.cuh"
//ʵ��__global__ void CUDAPulseDopplerRadarDisplayAlgorithmCore()�˺���
__host__ __device__ int CalculationOfOffset(int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int Axis_Y, int Axis_Z)
{
	return Axis_Y * Count_Of_Sampling_Point_Of_Monopulse + Axis_Z * Count_Of_Sampling_Point_Of_Monopulse * Count_Of_Pulse_Group;
}
__host__ __device__ float2 * GetArrayOneDimensionFloat2Data(float2*Initial_Position, int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int Axis_Y, int Axis_Z)
{
	float2 *One_Dimension_Float2_Data_Position = NULL;
	One_Dimension_Float2_Data_Position = Initial_Position + CalculationOfOffset(Count_Of_Sampling_Point_Of_Monopulse, Count_Of_Pulse_Group, Axis_Y, Axis_Z);
	return One_Dimension_Float2_Data_Position;
}
void CUDAPulseDopplerRadarDisplayAlgorithmCore(hipfftHandle Plan, float2 * Radar_Data_Cube_Address, int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int Antenna_Chanel_Count)
{
	for (int Index = 0; Index<Antenna_Chanel_Count; Index++)
	{
		float2 * Radar_Signal_Padding = GetArrayOneDimensionFloat2Data(Radar_Data_Cube_Address, Count_Of_Sampling_Point_Of_Monopulse, Count_Of_Pulse_Group, 0, Index);
		checkCudaErrors(hipfftExecC2C(Plan, (hipfftComplex *)(Radar_Signal_Padding), (hipfftComplex *)(Radar_Signal_Padding), HIPFFT_FORWARD));
	}
}