#include "hip/hip_runtime.h"
#include "CUDABasebandReceiverAlgorithmCore.cuh"
#include "../CUDAHeaderFileOfHierarchy1DLL/CUDAMathematicsFoundationArithmeticCore.cuh"
__device__ __host__ double NoiseVoltage(double Power, double Impedance_Temperature)
{
	return sqrt(Power * Impedance_Temperature);
}

__device__ __host__ double NoisePower(double Band_Width, double Noise_Factor, double Impedance_Temperature)
{
	double Boltzmann_Constant = 1.3806505e-23;//������������k
	double Perfect_Receiver_Noise_Power = Boltzmann_Constant*Band_Width;//������ջ�������������
	double Holistic_Noise_Power = Noise_Factor * Perfect_Receiver_Noise_Power;//������������ϵ�������������
	return Holistic_Noise_Power;
}

__device__ __host__ double ReceiverNoiseVoltage(double Receiver_Gain, double Band_Width, double Noise_Factor, double Impedance, double Impedance_Temperature)
{
	double Band_Width_Temporary = Band_Width * 1.2 * 1.2;
	double Noise_Power = NoisePower(Band_Width_Temporary, Noise_Factor, Impedance_Temperature) * Receiver_Gain;
	double Noise_Voltage = NoiseVoltage(Noise_Power, Impedance);
	return Noise_Voltage;
}

__host__ __device__ int CalculationOfOffset(int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int Axis_Y, int Axis_Z)
{
	return Axis_Y * Count_Of_Sampling_Point_Of_Monopulse + Axis_Z * Count_Of_Sampling_Point_Of_Monopulse * Count_Of_Pulse_Group;
}

__host__ __device__ float2 * GetArrayOneDimensionFloat2Data(float2*Initial_Position, int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int Axis_Y, int Axis_Z)
{
	float2 *One_Dimension_Float2_Data_Position = NULL;
	One_Dimension_Float2_Data_Position = Initial_Position + CalculationOfOffset(Count_Of_Sampling_Point_Of_Monopulse, Count_Of_Pulse_Group, Axis_Y, Axis_Z);
	return One_Dimension_Float2_Data_Position;
}

__global__ void SensitivityTimeControlEcho(float2 * Radar_Data_Cube_Address, int Total_Size, float Sensitivity_Time_Control_Coefficient, float Pulse_Repetition_Time)
{
	float Time_Step = Pulse_Repetition_Time / (Total_Size*1.0f);
	int Total_Threads = gridDim.x * blockDim.x;
	int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int Index = Thread_ID; Index < Total_Size; Index += Total_Threads)
	{
		float Time_Step_Temporary = (Index + 1.0)*Time_Step;
		float STC_Attenuation = pow(Time_Step_Temporary / Pulse_Repetition_Time, Sensitivity_Time_Control_Coefficient);
		Radar_Data_Cube_Address[Index].x = (Radar_Data_Cube_Address[Index].x * STC_Attenuation);
		Radar_Data_Cube_Address[Index].y = (Radar_Data_Cube_Address[Index].y * STC_Attenuation);
	}
}

__global__ void OverLapSignal(float2 *Total_Received_Signal, float2  *Random_Sigal, int Sampling_Point_Total_Count, float Amtitude)
{
	int Total_Threads = gridDim.x * blockDim.x;
	int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int Index = Thread_ID; Index < Sampling_Point_Total_Count; Index += Total_Threads)
	{
		Total_Received_Signal[Index].x = Total_Received_Signal[Index].x + Random_Sigal[Index].x*Amtitude;
		Total_Received_Signal[Index].y = Total_Received_Signal[Index].y + Random_Sigal[Index].y*Amtitude;
	}
}

__global__ void LogarithmicAmplifierEcho(float2 * Radar_Data_Cube_Address, int Total_Size, float Logarithmic_Amplifier_Voltage)//U01��ѹ
{
	int Total_Threads = gridDim.x * blockDim.x;
	int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
	for (unsigned int Index = Thread_ID; Index < Total_Size; Index += Total_Threads)
	{
		int sgnx;
		if ((Radar_Data_Cube_Address[Index].x) > 0)
		{
			sgnx = 1;

		}
		else
		{
			sgnx = -1;
		}
		float tmpx = fabsf(Radar_Data_Cube_Address[Index].x);
		if (tmpx > Logarithmic_Amplifier_Voltage)
		{
			Radar_Data_Cube_Address[Index].x = (Logarithmic_Amplifier_Voltage + Logarithmic_Amplifier_Voltage*log(tmpx / Logarithmic_Amplifier_Voltage))*sgnx;
		}


		int sgny;
		if ((Radar_Data_Cube_Address[Index].y) > 0)
		{
			sgny = 1;

		}
		else
		{
			sgny = -1;
		}
		float tmpy = fabsf(Radar_Data_Cube_Address[Index].y);
		if (tmpy > Logarithmic_Amplifier_Voltage)
		{
			Radar_Data_Cube_Address[Index].y = (Logarithmic_Amplifier_Voltage + Logarithmic_Amplifier_Voltage*log(tmpy / Logarithmic_Amplifier_Voltage))*sgny;
		}
	}
}

__global__ void AmplifierSaturation(float2 * Radar_Data_Cube_Address, int Total_Size, float Voltage_Saturation, float Voltage_Max, float Saturation_Factor)
{
	int Total_Threads = gridDim.x * blockDim.x;
	int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
	for (unsigned int Index = Thread_ID; Index < Total_Size; Index += Total_Threads)
	{
		int sgnx;
		if ((Radar_Data_Cube_Address[Index].x) > 0)
		{
			sgnx = 1;
		}
		else
		{
			sgnx = -1;
		}
		float tmpx = fabsf(Radar_Data_Cube_Address[Index].x);
		if (tmpx > Voltage_Saturation)
		{
			Radar_Data_Cube_Address[Index].x = (Voltage_Max - pow(Saturation_Factor, (tmpx - Voltage_Saturation)))*sgnx;
		}


		int sgny;
		if ((Radar_Data_Cube_Address[Index].y) > 0)
		{
			sgny = 1;
		}
		else
		{
			sgny = -1;
		}
		float tmpy = fabsf(Radar_Data_Cube_Address[Index].y);
		if (tmpy > Voltage_Saturation)
		{
			Radar_Data_Cube_Address[Index].y = (Voltage_Max - pow(Saturation_Factor, (tmpy - Voltage_Saturation)))*sgny;
		}
	}
}

__global__ void ADConverter(float2 * Radar_Data_Cube_Address, int Total_Size, float Min_Voltage, float Max_Voltage, float Delta_Voltage)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	for (unsigned int Index = threadID; Index < Total_Size; Index += numThreads)
	{
		if (Radar_Data_Cube_Address[Index].x <= Min_Voltage)
		{
			Radar_Data_Cube_Address[Index].x = Min_Voltage;
		}
		else
		{
			if (Radar_Data_Cube_Address[Index].x >= Max_Voltage)
			{
				Radar_Data_Cube_Address[Index].x = Max_Voltage;
			}
			else
			{
				Radar_Data_Cube_Address[Index].x = round((Radar_Data_Cube_Address[Index].x - Min_Voltage) / Delta_Voltage)*Delta_Voltage - Max_Voltage;;
			}
		}

		if (Radar_Data_Cube_Address[Index].y <= Min_Voltage)
		{
			Radar_Data_Cube_Address[Index].y = Min_Voltage;
		}
		else
		{
			if (Radar_Data_Cube_Address[Index].y >= Max_Voltage)
			{
				Radar_Data_Cube_Address[Index].y = Max_Voltage;
			}
			else
			{
				Radar_Data_Cube_Address[Index].y = round((Radar_Data_Cube_Address[Index].y - Min_Voltage) / Delta_Voltage)*Delta_Voltage - Max_Voltage;;
			}
		}
	}
}

void CUDABasebandReceiverAlgorithmCore(float2 * Radar_Data_Cube_Address, float2 * Receiver_Noise_Data_Address, int Total_Size_Of_Receiver_Noise_Data, int BlockSize, double Receiver_Gain, double Band_Width, double Noise_Factor, double Impedance, double Impedance_Temperature, float Sensitivity_Time_Control_Coefficient, double Window_Of_Range_Right, double Window_Of_Range_Left, double Logarithmic_Amplifier_Voltage, float Voltage_Saturation, float Max_Voltage, float Saturation_Factor, int ADC_Number, float Pulse_Repetition_Time, int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int Antenna_Chanel_Count, bool Sensitivity_Time_Control, bool Use_Window_Of_Range, bool Automatic_Gain_Control, bool Logarithmic_Amplifier, bool Amplifier_Saturation, bool Analog_Digital_Conversion)
{
	hiprandGenerator_t Random_Generator;
	hiprandRngType_t Random_Type = HIPRAND_RNG_PSEUDO_MTGP32;
	hiprandStatus_t Curand_Status= hiprandCreateGenerator(&Random_Generator, Random_Type);
	double Amptitude = ReceiverNoiseVoltage(Receiver_Gain, Band_Width, Noise_Factor, Impedance, Impedance_Temperature);
	hiprandStatus_t Curand_Result = hiprandGenerateNormal(Random_Generator, (float*)Receiver_Noise_Data_Address, Total_Size_Of_Receiver_Noise_Data * 2, 0.0, pow(Amptitude, 2)*Impedance);
	int GridSize = (Count_Of_Sampling_Point_Of_Monopulse + BlockSize - 1) / BlockSize;
	for (int Index_Z = 0; Index_Z < Antenna_Chanel_Count; Index_Z++)
	{
		for (int Index_Y = 0; Index_Y < Count_Of_Pulse_Group; Index_Y++)
		{
			float2 * Radar_Signal_Padding = GetArrayOneDimensionFloat2Data(Radar_Data_Cube_Address, Count_Of_Sampling_Point_Of_Monopulse, Count_Of_Pulse_Group, Index_Y, Index_Z);
			if (Sensitivity_Time_Control)
			{
				SensitivityTimeControlEcho << <GridSize , BlockSize>> >(Radar_Signal_Padding, Count_Of_Sampling_Point_Of_Monopulse, Sensitivity_Time_Control_Coefficient, Pulse_Repetition_Time);
				if (Use_Window_Of_Range)
				{
					double Window_Of_Range_Size = 2 * (Window_Of_Range_Right - Window_Of_Range_Left) / 3e8;
					SensitivityTimeControlEcho << <GridSize, BlockSize >> >(Radar_Signal_Padding, Count_Of_Sampling_Point_Of_Monopulse, Sensitivity_Time_Control_Coefficient, Window_Of_Range_Size);
				}
			}
			float2 * Receiver_Thermal_Noise_Signal_Data = GetArrayOneDimensionFloat2Data(Receiver_Noise_Data_Address, Count_Of_Sampling_Point_Of_Monopulse, Count_Of_Pulse_Group, Index_Y, Index_Z);
			OverLapSignal << <GridSize, BlockSize >> >(Radar_Signal_Padding, Receiver_Thermal_Noise_Signal_Data, Count_Of_Sampling_Point_Of_Monopulse, 1.0);
			if (Automatic_Gain_Control)
			{
				//���㷨��û�п�����
			}
			if (Logarithmic_Amplifier)
			{
				LogarithmicAmplifierEcho << <GridSize, BlockSize >> >(Radar_Signal_Padding, Count_Of_Sampling_Point_Of_Monopulse, Logarithmic_Amplifier_Voltage);

			}
			if (Amplifier_Saturation)
			{
				AmplifierSaturation << <GridSize, BlockSize >> >(Radar_Signal_Padding, Count_Of_Sampling_Point_Of_Monopulse, Voltage_Saturation, Max_Voltage, Saturation_Factor);

			}
			if (Analog_Digital_Conversion)
			{
				ADConverter << <GridSize, BlockSize >> >(Radar_Signal_Padding, Count_Of_Sampling_Point_Of_Monopulse, -1.0 * Max_Voltage, 1.0 * Max_Voltage, 2.0* Max_Voltage / pow(2.0, ADC_Number));

			}
		}
	}


}