#include "hip/hip_runtime.h"
#include "CUDAEchoSignalGeneratorAlgorithmCore.cuh"
#include "../CUDAHeaderFileOfHierarchy1DLL/CUDAMathematicsFoundationArithmeticCore.cuh"
#define  PI_CUDA 3.141592653589793f
#define PI 3.1415926535
__device__ __host__ double LinearFrequencyModulation(double Frequency_Begin, double Slope_Of_Frequency_Modulation, double Time)
{
	double Delta_Frequency = Slope_Of_Frequency_Modulation * Time;
	double Frequency_Slope = Frequency_Begin + Delta_Frequency;
	return Frequency_Slope;
}

__device__ __host__ double NonlinearityFrequencyModulationTan(double Frequency_Begin, double Band_Width, double Pulse_Width, double Time)
{
	double Alpha = 2.5;
	double Beta = atan(Alpha);
	double Delta_Frequency = Band_Width *tan(2.0 * Beta * (Time) / Pulse_Width) / (2.0 * tan(Beta));
	double Frequency_Slope = Frequency_Begin + Delta_Frequency;
	return Frequency_Slope;
}

__device__ __host__ double NonlinearityFrequencyModulationPolynomial(double Frequency_Begin, double Band_Width, double Pulse_Width, double Time)
{
	//���ڶ���ʽ�ķ����Ե�Ƶ
	double Polynomial_Parameter[7] = { -0.1145, 0.0396, -0.0202, 0.0118, 0.0082, 0.0055, -0.0040 };
	double Temporary = 0;
	for (int i = 0; i < 7; i++)
	{
		Temporary = Temporary + Polynomial_Parameter[i] * sinf(2 * PI_CUDA*(i + 1)*Time / Pulse_Width);//sinf�����Ƿ���cuda�еĺ���
	}
	double Delta_Frequency = Band_Width * (Time / Pulse_Width + Temporary);
	double Frequency_Slope = Frequency_Begin + Delta_Frequency;
	return Frequency_Slope;
}

__device__ __host__ double TwoPhaseBarkerOne()
{
	double Phase = 0.0;
	return Phase;
}
__device__ __host__ double TwoPhaseBarkerTwo(double Pulse_Width, double Time)
{
	int Barker_Value = Time / (Pulse_Width / 2.0);
	double Phase;
	if (Barker_Value == 1)
	{
		Phase = PI_CUDA;
	}
	else
	{
		Phase = 0;
	}
	return Phase;
}

__device__ __host__ double ThreePhaseBarker(double Pulse_Width, double Time)
{
	int Barker_Value = Time / (Pulse_Width / 3.0);
	double Phase;
	if (Barker_Value == 2)
	{
		Phase = PI_CUDA;
	}
	else
	{
		Phase = 0;
	}
	return Phase;
}

__device__ __host__ double FourPhaseBarkerOne(double Pulse_Width, double Time)
{
	int Barker_Value = Time / (Pulse_Width / 4.0);
	double Phase;
	if (Barker_Value == 2)
	{
		Phase = PI_CUDA;
	}
	else
	{
		Phase = 0;
	}
	return Phase;
}

__device__ __host__ double FourPhaseBarkerTwo(double Pulse_Width, double Time)
{
	int Barker_Value = Time / (Pulse_Width / 4.0);
	double Phase;
	if (Barker_Value == 3)
	{
		Phase = PI_CUDA;
	}
	else
	{
		Phase = 0;
	}
	return Phase;
}

__device__ __host__ double FivePhaseBarker(double Pulse_Width, double Time)
{
	int Barker_Value = Time / (Pulse_Width / 5.0);
	double Phase;
	if (Barker_Value == 3)
	{
		Phase = PI_CUDA;
	}
	else
	{
		Phase = 0;
	}
	return Phase;
}

__device__ __host__ double SevenPhaseBarker(double Pulse_Width, double Time)
{
	int Barker_Value = Time / (Pulse_Width / 7.0);
	double Phase;
	if (Barker_Value == 3 || Barker_Value == 4 || Barker_Value == 6)
	{
		Phase = PI_CUDA;
	}
	else
	{
		Phase = 0;
	}
	return Phase;
}

__device__ __host__ double ElevenPhaseBarker(double Pulse_Width, double Time)
{
	int Barker_Value = Time / (Pulse_Width / 11.0);
	double Phase;
	if (Barker_Value == 3 || Barker_Value == 4 || Barker_Value == 5 || Barker_Value == 7 || Barker_Value == 8 || Barker_Value == 10)
	{
		Phase = PI_CUDA;
	}
	else
	{
		Phase = 0;
	}
	return Phase;
}

__device__ __host__ double ThirteenPhaseBarker(double Pulse_Width, double Time)
{
	int Barker_Value = Time / (Pulse_Width / 13.0);
	double Phase;
	if ((Barker_Value == 5) || (Barker_Value == 6) || (Barker_Value == 9) || (Barker_Value == 11))
	{
		Phase = PI_CUDA;
	}
	else
	{
		Phase = 0;
	}
	return Phase;
}

__device__ __host__ double PhaseBarkerMN(double Pulse_Width, double Time, int M, int N, int Index_First, int Index_Second)
{
	int Barker2One[2] = { 1, 1 };
	int Barker2Two[2] = { 1, -1 };
	int Barker3[3] = { 1, 1, -1 };
	int Barker4One[4] = { 1, 1, -1, 1 };
	int Barker4Two[4] = { 1, 1, 1, -1 };
	int Barker5[5] = { 1, 1, 1, -1, 1 };
	int Barker7[7] = { 1, 1, 1, -1, -1, 1, -1 };
	int Barker11[11] = { 1, 1, 1, -1, -1, -1, 1, -1, -1, 1, -1 };
	int Barker13[13] = { 1, 1, 1, 1, 1, -1, -1, 1, 1, -1, 1, -1, 1 };

	std::vector < std::vector<int> > Total_Barker_Code;

	for (int i = 0; i < 9; i++)
	{
		if (i == 0)
		{
			std::vector<int> BarkerCode(Barker2One, Barker2One + sizeof(Barker2One) / sizeof(Barker2One[0]));
			Total_Barker_Code.push_back(BarkerCode);
		}
		else if (i == 1)
		{
			std::vector<int> BarkerCode(Barker2Two, Barker2Two + sizeof(Barker2Two) / sizeof(Barker2Two[0]));
			Total_Barker_Code.push_back(BarkerCode);
		}
		else if (i == 2)
		{
			std::vector<int> BarkerCode(Barker3, Barker3 + sizeof(Barker3) / sizeof(Barker3[0]));
			Total_Barker_Code.push_back(BarkerCode);
		}
		else if (i == 3)
		{
			std::vector<int> BarkerCode(Barker4One, Barker4One + sizeof(Barker4One) / sizeof(Barker4One[0]));
			Total_Barker_Code.push_back(BarkerCode);
		}
		else if (i == 4)
		{
			std::vector<int> BarkerCode(Barker4Two, Barker4Two + sizeof(Barker4Two) / sizeof(Barker4Two[0]));
			Total_Barker_Code.push_back(BarkerCode);
		}
		else if (i == 5)
		{
			std::vector<int> BarkerCode(Barker5, Barker5 + sizeof(Barker5) / sizeof(Barker5[0]));
			Total_Barker_Code.push_back(BarkerCode);
		}
		else if (i == 6)
		{
			std::vector<int> BarkerCode(Barker7, Barker7 + sizeof(Barker7) / sizeof(Barker7[0]));
			Total_Barker_Code.push_back(BarkerCode);
		}
		else if (i == 7)
		{
			std::vector<int> BarkerCode(Barker11, Barker11 + sizeof(Barker11) / sizeof(Barker11[0]));
			Total_Barker_Code.push_back(BarkerCode);
		}
		else if (i == 8)
		{
			std::vector<int> BarkerCode(Barker13, Barker13 + sizeof(Barker13) / sizeof(Barker13[0]));
			Total_Barker_Code.push_back(BarkerCode);
		}

	}
	std::vector<int> Kornecker_Product;
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < N; j++)
		{
			int Scalar_Product = Total_Barker_Code[Index_First][i] * Total_Barker_Code[Index_Second][j];
			Kornecker_Product.push_back(Scalar_Product);
		}
	}
	int Barker_Value = Time / (Pulse_Width / Kornecker_Product.size());
	double Phase;
	if (Kornecker_Product[Barker_Value] == -1)
	{
		Phase = PI_CUDA;
	}
	else
	{
		Phase = 0;
	}
	return Phase;
}

__device__ __host__ double PhaseFrank(int M, double Pulse_Width, double Time)
{
	int N = M * M;

	double Phase;
	int Row = unsigned int(Time / (Pulse_Width / N)) / M + 1;
	int Column = unsigned int(Time / (Pulse_Width / N)) % M + 1;
	Phase = 2 * PI_CUDA / M * (Row - 1) * (Column - 1);
	return Phase;
}

__device__ __host__ double PhaseLKPOne(int M, double Pulse_Width, double Time)
{
	int N = M * M;

	double Phase;
	int Row = unsigned int(Time / (Pulse_Width / N)) / M + 1;
	int Column = unsigned int(Time / (Pulse_Width / N)) % M + 1;
	Phase = -PI_CUDA / (1.0*M)*(M - 2 * Row + 1.0)*(M*(Column - 1.0) + Column - 1.0);
	return Phase;
}

__device__ __host__ double PhaseLKPTwo(int M, double Pulse_Width, double Time)
{
	int N = M * M;
	double Phase;
	int Row = unsigned int(Time / (Pulse_Width / N)) / M + 1;
	int Column = unsigned int(Time / (Pulse_Width / N)) % M + 1;
	Phase = -PI_CUDA / 2.0 / (1.0*M) * (2.0* Row - 1.0 - M)*(2.0* Column - 1.0 - M);
	return Phase;
}

__device__ __host__ double PhaseLKPThree(int N, double Pulse_Width, double Time)
{
	double Phase;
	int Chip_Number = Time / (Pulse_Width / N);
	Phase = PI_CUDA / N*Chip_Number*Chip_Number;
	return Phase;
}

__device__ __host__ double PhaseLKPFour(int N, double Pulse_Width, double Time)
{
	double Phase;
	unsigned int Chip_Number = Time / (Pulse_Width / N);
	Phase = PI_CUDA / N*Chip_Number*Chip_Number - PI_CUDA*Chip_Number;
	return Phase;
}

__device__ __host__ double GetPowerSource(double Power, double Impedance)
{
	return sqrt(Power * Impedance);
}

__device__ __host__ double GetTransmitPower(double Transmit_Power, double Antenna_Transmit_Gain, double Antenna_Receive_Gain, double Wave_Length, double RCS, double Distance)
{
	double Pr;
	Pr = Transmit_Power*Antenna_Transmit_Gain*Antenna_Receive_Gain*Wave_Length*Wave_Length*RCS / (pow((4 * PI), 3.0) * pow(Distance, 4.0));
	return Pr;
}

__device__ __host__ double GetPhaseZero(double Medium_Frequency, double Current_Time)
{
	double Tmporary = Medium_Frequency*Current_Time - (unsigned int)(Medium_Frequency*Current_Time);
	double Phase_Zero = 2.0 * PI*Tmporary;
	return Phase_Zero;
}

__device__ __host__ double GetDelta(double Current_Distance_Between_Target_and_Radar, bool Is_Two_Way)
{
	if (Is_Two_Way)
	{
		return Current_Distance_Between_Target_and_Radar / 299792458.0 * 2.0;
	}
	else
	{
		return Current_Distance_Between_Target_and_Radar / 299792458.0;
	}
}

__device__ __host__ float GetPulseBasisOfPRTList(float * Pulse_Transmit_Time_List, int Pulse_ID)
{
	float Pulse_Time = Pulse_Transmit_Time_List[Pulse_ID];
	return Pulse_Time;
}

__device__ __host__ double GetPulse(double PRT, int Pulse_ID)
{
	return PRT * Pulse_ID;
}

__global__ void GenerateOneEchoComplex(int Type, float Phase, float Frequency_Begin, float Frequency_End, float Pulse_Width, int Overlap_Position, int End_Position, float Amplitude, int Total_Size, float2* Complex_Data)
{
	int Total_Threads = gridDim.x * blockDim.x;
	int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
	float Band_Width = Frequency_End - Frequency_Begin;
	int Pulse_Size = End_Position - Overlap_Position;
	float TimeStep = Pulse_Width / (Pulse_Size*1.0);
	float FreqSlope = Band_Width / Pulse_Width / 2.0;
	float PI_Double = 2.0 * PI_CUDA;
	float PhaseRT = 0.0;
	float FreqRT = (Frequency_End + Frequency_Begin) / 2.0;
	for (int Index = Thread_ID; Index < Pulse_Size; Index += Total_Threads)
	{
		int Array_Index = Index + Overlap_Position;
		if ((Array_Index) < Total_Size)
		{

			float TimeRT = TimeStep*(Index);
			switch (Type)
			{
			case 0:
			{
				FreqRT = LinearFrequencyModulation(Frequency_Begin, FreqSlope, TimeRT);
				break;
			}
			case 1:
			{
				FreqRT = NonlinearityFrequencyModulationTan(Frequency_Begin, Band_Width, Pulse_Width, TimeRT);
				break;
			}
			case 2:
			{
				FreqRT = NonlinearityFrequencyModulationPolynomial(Frequency_Begin, Band_Width, Pulse_Width, TimeRT);
				break;
			}
			case 3:
			{
				PhaseRT = TwoPhaseBarkerOne();
				break;
			}
			case 4:
			{
				PhaseRT = TwoPhaseBarkerTwo(Pulse_Width, TimeRT);
				break;
			}
			case 5:
			{
				PhaseRT = ThreePhaseBarker(Pulse_Width, TimeRT);
				break;
			}
			case 6:
			{
				PhaseRT = FourPhaseBarkerOne(Pulse_Width, TimeRT);
				break;
			}
			case 7:
			{
				PhaseRT = FourPhaseBarkerTwo(Pulse_Width, TimeRT);
				break;
			}
			case 8:
			{
				PhaseRT = FivePhaseBarker(Pulse_Width, TimeRT);
				break;
			}
			case 9:
			{
				PhaseRT = SevenPhaseBarker(Pulse_Width, TimeRT);
				break;
			}
			case 10:
			{
				PhaseRT = ElevenPhaseBarker(Pulse_Width, TimeRT);
				break;
			}
			case 11:
			{
				PhaseRT = ThirteenPhaseBarker(Pulse_Width, TimeRT);
				break;
			}
			case 12:
			{
				PhaseRT = PhaseBarkerMN(Pulse_Width, TimeRT, 4, 5, 4, 5);
				break;
			}
			case 13:
			{
				FreqRT = LinearFrequencyModulation(Frequency_Begin, FreqSlope, TimeRT);
				PhaseRT = ElevenPhaseBarker(Pulse_Width, TimeRT);
				break;
			}
			case 14:
			{
				PhaseRT = PhaseFrank(8, Pulse_Width, TimeRT);
				break;
			}
			case 15:
			{
				PhaseRT = PhaseLKPOne(8, Pulse_Width, TimeRT);
				break;
			}
			case 16:
			{
				PhaseRT = PhaseLKPTwo(8, Pulse_Width, TimeRT);
				break;
			}
			case 17:
			{
				PhaseRT = PhaseLKPThree(64, Pulse_Width, TimeRT);
				break;
			}
			case 18:
			{
				PhaseRT = PhaseLKPFour(64, Pulse_Width, TimeRT);
				break;
			}
			}
			float Echo_Signal_X = cos(PI_Double*FreqRT*(TimeRT - Pulse_Width / 2) + Phase + PhaseRT)*Amplitude;
			float Echo_Signal_Y = sin(PI_Double*FreqRT*(TimeRT - Pulse_Width / 2) + Phase + PhaseRT)*Amplitude;
			Complex_Data[Array_Index].x += Echo_Signal_X;
			Complex_Data[Array_Index].y += Echo_Signal_Y;
		}
	}
}

__host__ __device__ int CalculationOfOffset(int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int Axis_Y, int Axis_Z)
{
	return Axis_Y * Count_Of_Sampling_Point_Of_Monopulse + Axis_Z * Count_Of_Sampling_Point_Of_Monopulse * Count_Of_Pulse_Group;
}

__host__ __device__ float2 * GetArrayOneDimensionFloat2Data(float2*Initial_Position, int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int Axis_Y, int Axis_Z)
{
	float2 *One_Dimension_Float2_Data_Position = NULL;
	One_Dimension_Float2_Data_Position = Initial_Position + CalculationOfOffset(Count_Of_Sampling_Point_Of_Monopulse, Count_Of_Pulse_Group, Axis_Y, Axis_Z);
	return One_Dimension_Float2_Data_Position;
}


void CUDAEchoSignalGeneratorAlgorithmCore(float2 * Radar_Float2_Data_Cube_Address, int BlockSize, int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int Antenna_Chanel_Count, double Signal_Transmit_Time, double Distance_Between_Target_and_Radar, double Target_Velocity_Relative_to_Radar, double RCS, double Receiver_Gain, double Transmit_Power, double Antenna_Transmit_Gain, double Antenna_Receive_Gain, double Wave_Length, double Impedance, float Baseband_Sampling_Frequency, float Pulse_Width, float Band_Width, float PRT, float *PRT_List, int PRT_List_Size, int Type)
{
	int GridSize = (Baseband_Sampling_Frequency * Pulse_Width + BlockSize - 1) / BlockSize;
	for (int Index_Y = 0; Index_Y < Count_Of_Pulse_Group; Index_Y++)
	{
		double Current_Distance_Between_Target_and_Radar = 0.0;//Ŀ�굱ǰ����
		float Pulse = 0.0;
		if (PRT_List_Size <= 0)
		{
			Pulse = GetPulse(PRT, Index_Y);
		}
		else
		{
			Pulse = GetPulseBasisOfPRTList(PRT_List, Index_Y);
		}
		Current_Distance_Between_Target_and_Radar = Distance_Between_Target_and_Radar + Target_Velocity_Relative_to_Radar * Pulse;
		double Delta = GetDelta(Current_Distance_Between_Target_and_Radar, true);//Ϊʲô��ֱ����GetDelta������д��Ϊtrue״̬��
		double Current_Time = Signal_Transmit_Time + Delta;
		double Pulse_Zero = GetPhaseZero(3.0e8 / Wave_Length, Current_Time);
		double Receiver_Power = GetTransmitPower(Transmit_Power, Antenna_Transmit_Gain, Antenna_Receive_Gain, Wave_Length, RCS, Current_Distance_Between_Target_and_Radar);
		double Power_Source_Value = GetPowerSource(Receiver_Power, Impedance);
		int Overlap_Position = Baseband_Sampling_Frequency * Delta;
		int End_Position = Overlap_Position + Baseband_Sampling_Frequency * Pulse_Width;
		int Pulse_Size = End_Position - Overlap_Position;
		float Frequency_Begin = -Band_Width / 2;
		float Frequency_End = Band_Width / 2;
		for (int Index_Z = 0; Index_Z < Antenna_Chanel_Count; Index_Z++)
		{
			float2 * Radar_Signal_Padding = GetArrayOneDimensionFloat2Data(Radar_Float2_Data_Cube_Address, Count_Of_Sampling_Point_Of_Monopulse, Count_Of_Pulse_Group, Index_Y, Index_Z);
			GenerateOneEchoComplex << <GridSize, BlockSize >> >(Type, Pulse_Zero, Frequency_Begin, Frequency_End, Pulse_Width, Overlap_Position, End_Position, Power_Source_Value, Count_Of_Sampling_Point_Of_Monopulse, Radar_Signal_Padding);
		}
	}

}