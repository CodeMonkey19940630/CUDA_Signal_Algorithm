#include "hip/hip_runtime.h"
#include "CUDAOneDimensionEchoExtractionAlgorithmCore.cuh"
#include "../CUDAHeaderFileOfHierarchy1DLL/CUDAMathematicsFoundationArithmeticCore.cuh"

__device__ __forceinline__ float AtomicMinFloat(float* addr, float value) {
	float old;
	old = !signbit(value) ? __int_as_float(atomicMin((int*)addr, __float_as_int(value))) :
		__uint_as_float(atomicMax((unsigned int*)addr, __float_as_uint(value)));

	return old;
}

__device__ __forceinline__ float AtomicMaxFloat(float* addr, float value) {
	float old;
	old = !signbit(value) ? __int_as_float(atomicMax((int*)addr, __float_as_int(value))) :
		__uint_as_float(atomicMin((unsigned int*)addr, __float_as_uint(value)));

	return old;
}

__global__ void FindMax(float *Max_Temporary_Index, float *CFAR_Input_Absolute_Address, int Begin_Index, int End_Index)
{
	int Thread_ID = blockDim.x * blockIdx.x + threadIdx.x;
	if ((Thread_ID >= Begin_Index) && (Thread_ID <= End_Index))//ֻ��һ�������ڵ��̷߳��ϸ�Ҫ��
	{
		AtomicMaxFloat(Max_Temporary_Index, CFAR_Input_Absolute_Address[Thread_ID]);
	}
}

__global__ void EchoCollection(int *Echo_Begin_End_Index_Record, bool *CFAR_Result_Zero_Padding, int Count_Of_Sampling_Point_Of_Monopulse)
{
	int Thread_ID = blockDim.x * blockIdx.x + threadIdx.x;
	if (Thread_ID < Count_Of_Sampling_Point_Of_Monopulse)
	{
		if ((CFAR_Result_Zero_Padding[Thread_ID] == 0) && (CFAR_Result_Zero_Padding[Thread_ID + 1] == 1))
		{
			Echo_Begin_End_Index_Record[Thread_ID] = Thread_ID + 1;
		}
		if ((CFAR_Result_Zero_Padding[Thread_ID] == 1) && (CFAR_Result_Zero_Padding[Thread_ID + 1] == 0))
		{
			Echo_Begin_End_Index_Record[Thread_ID] = Thread_ID;
		}

	}
}

void CUDAOneDimensionEchoExtractionAlgorithmCore(std::vector<std::pair<int, int>> &Echo_Begin_End_Index_Pair_Result, std::vector<std::pair<int, float>> &Max_Result, bool *CFAR_Result_Address, float *CFAR_Input_Absolute_Address, int Count_Of_Sampling_Point_Of_Monopulse, int BlockSize)
{
	//��������
	//CFAR_Result_Address��Ϊ0��1����
	//CFAR_Input_Absolute_Address��Ϊ��ʵֵ����
	//Count_Of_Sampling_Point_Of_MonopulseΪ�������еĳ���
	//BlockSize�鳤��
	//��Ҫ��CFAR_Result_Address���е�����ĩβ��0��������Count_Of_Sampling_Point_Of_Monopulse��BlockSize���¼�������GPU�Դ��ַ����CFAR_Result_Address�����ݸ��ƽ�����
	int GridSize = (Count_Of_Sampling_Point_Of_Monopulse / BlockSize) + 1;

	bool *CFAR_Result_Zero_Padding;//GPU
	int *Echo_Begin_End_Index_Record_GPU;//GPU
	int *Echo_Begin_End_Index_Record_CPU;//CPU
	checkCudaErrors(hipMalloc((void **)&CFAR_Result_Zero_Padding, GridSize * BlockSize * sizeof(bool)));//GPU
	checkCudaErrors(hipMalloc((void **)&Echo_Begin_End_Index_Record_GPU, GridSize * BlockSize * sizeof(int)));//GPU
	Echo_Begin_End_Index_Record_CPU = (int *)malloc(GridSize * BlockSize * sizeof(int));//CPU

	checkCudaErrors(hipMemset(CFAR_Result_Zero_Padding, 0, GridSize * BlockSize * sizeof(bool)));//���µ�GPU�Դ��ַ���г�ʼ��Ϊ0��
	checkCudaErrors(hipMemset(Echo_Begin_End_Index_Record_GPU, 0, GridSize * BlockSize * sizeof(int)));//���µ�GPU�Դ��ַ���г�ʼ��Ϊ0��
	memset(Echo_Begin_End_Index_Record_CPU, 0, GridSize * BlockSize * sizeof(int));//CPU

	checkCudaErrors(hipMemcpy(CFAR_Result_Zero_Padding, CFAR_Result_Address, Count_Of_Sampling_Point_Of_Monopulse * sizeof(bool), hipMemcpyDeviceToDevice));//��GPU��CFAR_Result_Address���ݸ��Ƶ�GPU��CFAR_Result_Zero_Padding��
	EchoCollection << <GridSize, BlockSize >> >(Echo_Begin_End_Index_Record_GPU, CFAR_Result_Zero_Padding, Count_Of_Sampling_Point_Of_Monopulse);//�ز�ʶ��
	//ʹ��hipMemcpy����ʽͬ��������Ż�Ϊ��ҳ�ڴ潫�˻�ʧЧ��
	checkCudaErrors(hipMemcpy(Echo_Begin_End_Index_Record_CPU, Echo_Begin_End_Index_Record_GPU, GridSize * BlockSize * sizeof(int), hipMemcpyDeviceToHost));//��Echo_Begin_End_Index_Record���ݸ��ƻ������ˣ������Echo_Begin_End_Index_Record_List�С�
	//��CPU��ѹ��Echo_Begin_End_Index_Record_CPU��
	std::vector<int> Echo_Begin_End_Index_Record_Compress_List;
	for (int Index = 0; Index < GridSize * BlockSize; Index++)
	{
		if (Echo_Begin_End_Index_Record_CPU[Index] != 0)
		{
			Echo_Begin_End_Index_Record_Compress_List.push_back(Echo_Begin_End_Index_Record_CPU[Index]);
		}
	}
	//pair��⣬��Echo_Count�ǲ��Ǵ���0��ż��
	int Echo_Count = 0;
	Echo_Count = Echo_Begin_End_Index_Record_Compress_List.size();
	if (Echo_Count > 0)
	{
		if ((Echo_Count % 2) != 0)
		{
			Echo_Begin_End_Index_Record_Compress_List.insert(Echo_Begin_End_Index_Record_Compress_List.begin(), 0);//ֻ���׵�ַλ�õ�����Ϊ1ʱ���п���Ϊ������
			Echo_Count = Echo_Count + 1;
		}
		if (Echo_Count > 0)
		{	//��ʼ�����ɶԷų���
			for (auto Index = 0; Index < Echo_Count; Index += 2)
			{
				Echo_Begin_End_Index_Pair_Result.push_back(std::make_pair(Echo_Begin_End_Index_Record_Compress_List[Index], Echo_Begin_End_Index_Record_Compress_List[Index + 1]));
				//����Ѱ�Ҵ�С�ĺ˺�����
				float * Max_Temporary_GPU;
				float *Max_Temporary_CPU;
				checkCudaErrors(hipMalloc((void **)&Max_Temporary_GPU, 1 * sizeof(float)));
				checkCudaErrors(hipMemset(Max_Temporary_GPU, 0, 1 * sizeof(float)));
				Max_Temporary_CPU = (float *)malloc(1 * sizeof(float));//CPU
				memset(Max_Temporary_CPU, 0, 1 * sizeof(float));//CPU
				FindMax << <GridSize, BlockSize >> >(Max_Temporary_GPU, CFAR_Input_Absolute_Address, Echo_Begin_End_Index_Record_Compress_List[Index], Echo_Begin_End_Index_Record_Compress_List[Index + 1]);
				checkCudaErrors(hipMemcpy(Max_Temporary_CPU, Max_Temporary_GPU, 1 * sizeof(float), hipMemcpyDeviceToHost));
				float Max_Value = 0.0;
				int Max_Index = 0;
				float * CFAR_Input_Absolute_Address_CPU = (float *)malloc(Count_Of_Sampling_Point_Of_Monopulse * sizeof(float));//CPU
				checkCudaErrors(hipMemcpy(CFAR_Input_Absolute_Address_CPU, CFAR_Input_Absolute_Address, Count_Of_Sampling_Point_Of_Monopulse * sizeof(float), hipMemcpyDeviceToHost));//��GPU��CFAR_Result_Address���ݸ��Ƶ�GPU��CFAR_Result_Zero_Padding��
				for (int Index = Echo_Begin_End_Index_Record_Compress_List[Index]; Index <= Echo_Begin_End_Index_Record_Compress_List[Index + 1]; Index++)
				{
					if (CFAR_Input_Absolute_Address_CPU[Index] > Max_Value)
					{
						Max_Value = CFAR_Input_Absolute_Address_CPU[Index];
						Max_Index = Index;
					}
				}
				Max_Result.push_back(std::make_pair(Max_Index, Max_Temporary_CPU[0]));
			}
		}
		else
		{
			//�����ж�����û�лز�
			printf("Still without any Echo!\n");
		}
	}
	else
	{
		//ѹ��û�лز�
		printf("Without any Echo!\n");
	}
}
