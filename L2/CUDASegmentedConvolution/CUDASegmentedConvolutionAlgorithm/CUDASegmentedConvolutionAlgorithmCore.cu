#include "hip/hip_runtime.h"
#include "CUDASegmentedConvolutionAlgorithmCore.cuh"
#include "../CUDAHeaderFileOfHierarchy1DLL/CUDAMathematicsFoundationArithmeticCore.cuh"

__global__ void Fliper(int Signal_Length, float2* Signal, float2* Signal_Padded)
{
	int Total_Threads = gridDim.x * blockDim.x;
	int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
	int j = 0;
	for (unsigned int i = Thread_ID; i < Signal_Length - 1; i += Total_Threads)
	{
		j = Signal_Length - i - 2;
		Signal_Padded[i] = Signal[j];
	}
}

__global__ void TranPosition(int Signal_Length, float2* Signal, int Signal_Padded_Length, float2* Signal_Padded)
{
	int Total_Threads = gridDim.x * blockDim.x;
	int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
	int j = 0;
	for (unsigned int i = Thread_ID; i < Signal_Length; i += Total_Threads)
	{
		j = i + Signal_Padded_Length - 1;
		Signal_Padded[i] = Signal[j];
	}
}


__global__ void TranPositionB(int Signal_Length, float2* Signal, int Signal_Padded_Length, float2* Signal_Padded)
{
	int Total_Threads = gridDim.x * blockDim.x;
	int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
	int j = 0;
	for (int i = Thread_ID; i < Signal_Length; i += Total_Threads)
	{
		j = i + Signal_Padded_Length - 1;
		Signal_Padded[i] = Signal[j];
	}
}

__global__ void TranPositionC(int Signal_Length, float2* Signal, int Signal_Padded_Length, float2* Signal_Padded, int FFT_Length)
{
	int Total_Threads = gridDim.x * blockDim.x;
	int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
	int j = 0;
	int k = 0;
	for (int i = Thread_ID; i < Signal_Length; i += Total_Threads)
	{
		j = i + Signal_Padded_Length - 1;
		k = FFT_Length - 1 - Signal_Length + i;
		Signal_Padded[j] = Signal[k];
	}
}

__global__ void TranPositionD(int Signal_Length, float2* Signal, int Signal_Padded_Length, float2* Signal_Padded)
{
	int Total_Threads = gridDim.x * blockDim.x;
	int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
	int j = 0;
	for (int i = Thread_ID; i < Signal_Length; i += Total_Threads)
	{
		j = Signal_Padded_Length + i;
		Signal_Padded[i] = Signal[j];
	}
}

__global__ void TranPositionE(int Signal_Length, float2* Signal, int Signal_Padded_Length, float2* Signal_Padded, int N3, int FFT_Length)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int j = 0;
	unsigned int k = 0;
	for (unsigned int i = threadID; i < Signal_Length; i += numThreads)
	{
		j = i + Signal_Padded_Length - 1;
		k = FFT_Length - 1 - N3 + i;
		Signal_Padded[j] = Signal[k];
	}
	return;

}

__global__ void TranPositionZ(int Signal_Length, float2* Signal, int Signal_Padded_Length, float2* Signal_Padded)
{
	int Total_Threads = gridDim.x * blockDim.x;
	int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
	int j = 0;
	for (int i = Thread_ID; i < Signal_Length; i += Total_Threads)
	{
		j = i + Signal_Padded_Length - 1;
		Signal_Padded[j] = Signal[i];
	}
}

int CalculateSectionLength(int Kernel_Length)
{
	// Calculate section length M that M + N - 1 is powers of 2 and M is close to N
	int FFT_Length = 1;
	while ((2 * Kernel_Length) > FFT_Length)
	{
		FFT_Length *= 2;
	}
	return FFT_Length;
}

void CUDASegmentedConvolutionAlgorithmCore(hipfftHandle Plan, int BlockSize, int Signal_Length, int Kernel_Length, float2* Signal, float2 *Kernel)
{
	int U = 0;
	hipfftComplex *Padded_Buffer, *Piece_Buffer, *Kernel_Buffer, *Kernel_Fliper;
	int FFT_Length = CalculateSectionLength(Kernel_Length);
	int Overlap_Length = FFT_Length - Kernel_Length;
	int	Convolution_Length = Signal_Length + Kernel_Length - 1;
	int Piece_count = (Convolution_Length + Overlap_Length - 1) / Overlap_Length;
	int The_Last = Convolution_Length%Overlap_Length;
	int Total_Length = 0;
	if (The_Last == 0){
		Total_Length = Signal_Length + Kernel_Length;
	}
	else{
		Total_Length = (Piece_count - 1)*Overlap_Length + FFT_Length;
	}
	checkCudaErrors(hipMalloc((void**)&Padded_Buffer, Total_Length * sizeof(hipfftComplex)));
	checkCudaErrors(hipMalloc((void**)&Piece_Buffer, FFT_Length * sizeof(hipfftComplex)));
	checkCudaErrors(hipMalloc((void**)&Kernel_Buffer, FFT_Length * sizeof(hipfftComplex)));
	checkCudaErrors(hipMalloc((void**)&Kernel_Fliper, Kernel_Length * sizeof(hipfftComplex)));

	checkCudaErrors(hipMemset(Padded_Buffer, 0, Total_Length * sizeof(hipfftComplex)));
	checkCudaErrors(hipMemset(Piece_Buffer, 0, FFT_Length * sizeof(hipfftComplex)));
	checkCudaErrors(hipMemset(Kernel_Buffer, 0, FFT_Length * sizeof(hipfftComplex)));
	checkCudaErrors(hipMemset(Kernel_Fliper, 0, Kernel_Length * sizeof(hipfftComplex)));

	int GridSize = (Signal_Length + BlockSize - 1) / BlockSize;
	TranPositionZ << <GridSize, BlockSize >> > (Signal_Length, Signal, Kernel_Length + 1, Padded_Buffer);
	GridSize = (Kernel_Length + BlockSize - 1) / BlockSize;
	Fliper << <GridSize, BlockSize >> >(Kernel_Length, Kernel, Kernel_Fliper);
	CUDAMathematicsFoundationArithmeticNamespace::ComplexConjugate << <GridSize, BlockSize >> >(Kernel_Fliper, Kernel_Length);
	TranPosition << <GridSize, BlockSize >> > (Kernel_Length, Kernel_Fliper, 1, Kernel_Buffer);
	hipfftExecC2C(Plan, Kernel_Buffer, Kernel_Buffer, HIPFFT_FORWARD);//����û��
	for (int Trans = 1; Trans <= ((Piece_count - 1)*Overlap_Length + 1); Trans += Overlap_Length)
	{
		GridSize = (FFT_Length + BlockSize - 1) / BlockSize;
		TranPositionB << <GridSize, BlockSize >> >(FFT_Length, Padded_Buffer, Trans, Piece_Buffer);
		hipfftExecC2C(Plan, Piece_Buffer, Piece_Buffer, HIPFFT_FORWARD);
		CUDAMathematicsFoundationArithmeticNamespace::ComplexPointwiseMultiplicationAndScale << <GridSize, BlockSize >> >(Piece_Buffer, Kernel_Buffer, FFT_Length, 1.0f / FFT_Length);
		hipfftExecC2C(Plan, Piece_Buffer, Piece_Buffer, HIPFFT_BACKWARD);
		if (U < Piece_count - 1)
		{
			GridSize = (Overlap_Length + BlockSize - 1) / BlockSize;
			TranPositionC << <GridSize, BlockSize >> > (Overlap_Length, Piece_Buffer, Trans, Signal, FFT_Length);
		}
		else
		{
			GridSize = (The_Last + BlockSize - 1) / BlockSize;
			TranPositionE << <GridSize, BlockSize >> >(The_Last, Piece_Buffer, Overlap_Length * (Piece_count - 1), Signal, Overlap_Length, FFT_Length);
		}
		U = U + 1;
	}
	GridSize = (Signal_Length + BlockSize - 1) / BlockSize;
	TranPositionD << <GridSize, BlockSize >> >(Signal_Length, Signal, Kernel_Length, Signal);
	checkCudaErrors(hipFree(Padded_Buffer));
	checkCudaErrors(hipFree(Piece_Buffer));
	checkCudaErrors(hipFree(Kernel_Buffer));
	checkCudaErrors(hipFree(Kernel_Fliper));
}