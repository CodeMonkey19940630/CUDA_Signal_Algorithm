#include "hip/hip_runtime.h"
#include "CUDAMovingTargetIndicatorAlgorithmCore.cuh"
#include "../CUDAHeaderFileOfHierarchy1DLL/CUDAMathematicsFoundationArithmeticCore.cuh"
__host__ __device__ int CalculationOfOffset(int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int Axis_Y, int Axis_Z)
{
	return Axis_Y * Count_Of_Sampling_Point_Of_Monopulse + Axis_Z * Count_Of_Sampling_Point_Of_Monopulse * Count_Of_Pulse_Group;
}

__host__ __device__ float2 * GetArrayOneDimensionFloat2Data(float2*Initial_Position, int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int Axis_Y, int Axis_Z)
{
	float2 *One_Dimension_Float2_Data_Position = NULL;
	One_Dimension_Float2_Data_Position = Initial_Position + CalculationOfOffset(Count_Of_Sampling_Point_Of_Monopulse, Count_Of_Pulse_Group, Axis_Y, Axis_Z);
	return One_Dimension_Float2_Data_Position;
}

__global__ void SecondOrderEigencanceller(float2* Pulse_Initial_Address_First, float2* Pulse_Initial_Address_Second,int Total_Size)
{
	int Total_Threads = gridDim.x * blockDim.x;
	int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int Index = Thread_ID; Index < Total_Size; Index += Total_Threads)
	{
		Pulse_Initial_Address_First[Index].x = Pulse_Initial_Address_First[Index].x - Pulse_Initial_Address_Second[Index].x;
		Pulse_Initial_Address_First[Index].y = Pulse_Initial_Address_First[Index].y - Pulse_Initial_Address_Second[Index].y;
	}
	__syncthreads();
}

__global__ void ThirdOrderEigencanceller(float2* Pulse_Initial_Address_First, float2* Pulse_Initial_Address_Second, float2* Pulse_Initial_Address_Third, int Total_Size)
{
	int Total_Threads = gridDim.x * blockDim.x;
	int Thread_ID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int Index = Thread_ID; Index < Total_Size; Index += Total_Threads)
	{
		Pulse_Initial_Address_First[Index].x = Pulse_Initial_Address_First[Index].x - 2.0 *  Pulse_Initial_Address_Second[Index].x + Pulse_Initial_Address_Third[Index].x;
		Pulse_Initial_Address_First[Index].y = Pulse_Initial_Address_First[Index].y - 2.0 *  Pulse_Initial_Address_Second[Index].y + Pulse_Initial_Address_Third[Index].y;
	}
	__syncthreads();
}

void CUDAMovingTargetIndicatorAlgorithmCore(float2 * Radar_Data_Cube_Address, int BlockSize, int Moving_Target_Indicator_Order, int Count_Of_Sampling_Point_Of_Monopulse, int Count_Of_Pulse_Group, int Antenna_Chanel_Count)
{
	if (Moving_Target_Indicator_Order >= 3)
	{
		int MTI_Order = 3;
		int GridSize = (Count_Of_Sampling_Point_Of_Monopulse + BlockSize - 1) / BlockSize;
		for (int Index_Z = 0; Index_Z < Antenna_Chanel_Count; Index_Z++)
		{
			for (int Index_Y = 0; Index_Y < Count_Of_Pulse_Group - MTI_Order + 1; Index_Y++)
			{
				float2 * Radar_Signal_Padding = GetArrayOneDimensionFloat2Data(Radar_Data_Cube_Address, Count_Of_Sampling_Point_Of_Monopulse, Count_Of_Pulse_Group, Index_Y, Index_Z);
				float2 * Pulse_Data_Initial_Address_First = Radar_Signal_Padding;
				float2 * Pulse_Data_Initial_Address_Second = Radar_Signal_Padding + Count_Of_Sampling_Point_Of_Monopulse;
				float2 * Pulse_Data_Initial_Address_Third = Radar_Signal_Padding + 2 * Count_Of_Sampling_Point_Of_Monopulse;
				ThirdOrderEigencanceller <<<GridSize, BlockSize>>>(Pulse_Data_Initial_Address_First, Pulse_Data_Initial_Address_Second, Pulse_Data_Initial_Address_Third, Count_Of_Sampling_Point_Of_Monopulse);
			}
		}
	}
	else if (Moving_Target_Indicator_Order >= 2 && Moving_Target_Indicator_Order < 3)
	{
		int MTI_Order = 2;
		int GridSize = (Count_Of_Sampling_Point_Of_Monopulse + BlockSize - 1) / BlockSize;
		for (int Index_Z = 0; Index_Z < Antenna_Chanel_Count; Index_Z++)
		{
			for (int Index_Y = 0; Index_Y < Count_Of_Pulse_Group - MTI_Order + 1; Index_Y++)
			{

				float2 * Radar_Signal_Padding = GetArrayOneDimensionFloat2Data(Radar_Data_Cube_Address, Count_Of_Sampling_Point_Of_Monopulse, Count_Of_Pulse_Group, Index_Y, Index_Z);
				float2 * Pulse_Data_Initial_Address_First = Radar_Signal_Padding;
				float2 * Pulse_Data_Initial_Address_Second = Radar_Signal_Padding + Count_Of_Sampling_Point_Of_Monopulse;
				SecondOrderEigencanceller <<<GridSize, BlockSize >>>(Pulse_Data_Initial_Address_First, Pulse_Data_Initial_Address_Second, Count_Of_Sampling_Point_Of_Monopulse);
			}
		}
	}
	else if (Moving_Target_Indicator_Order >= 1 && Moving_Target_Indicator_Order < 2)
	{
	}
}